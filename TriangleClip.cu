#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
#include "stdlib.h"
#include "stdio.h"
#include "clip.h"
#define EPS 0.00001

inline void CheckError(hipError_t error)
{
	if (error != hipSuccess)
	{
		printf("returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}
}

extern "C"
__host__ void initCUDA()
{
	int devID = 0;

//	if (checkCmdLineFlag(argc, (const char **)argv, "device"))
//	{
		//devID = getCmdLineArgumentInt(argc, (const char **)argv, "device");
		hipSetDevice(devID);
//	}

	hipError_t error;
	hipDeviceProp_t deviceProp;
	error = hipGetDevice(&devID);

	if (error != hipSuccess)
	{
		printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
	}

	error = hipGetDeviceProperties(&deviceProp, devID);

	if (deviceProp.computeMode == hipComputeModeProhibited)
	{
		fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
		exit(EXIT_SUCCESS);
	}

	if (error != hipSuccess)
	{
		printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
	}
	else
	{
		printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
	}
}

extern "C"
__host__ void loadDataToDevice(float* trgl_s, float* trgl_c, int ntrgl, int *pair, int npar)
{
    float *d_trgl_s;
    float *d_trgl_c;
    int2 *d_pair;
    float *d_clipped_vert;
    float *d_clipped_n_vert;

    hipError_t error;
    unsigned int mem_size = ntrgl * 6 * sizeof(float);//3 vertices, each vertex has x and y(2 float)

    error = hipMalloc((void **) &d_trgl_s, mem_size);
    CheckError(error);

    error = hipMalloc((void **) &d_trgl_c, mem_size);
    CheckError(error);

    error = hipMemcpy(d_trgl_s, trgl_s, mem_size, hipMemcpyHostToDevice);
    CheckError(error);

    error = hipMemcpy(d_trgl_c, trgl_c, mem_size, hipMemcpyHostToDevice);
    CheckError(error);

    unsigned int mem_size_pair = npar * 2 * sizeof(int);

    error = hipMalloc((void **) &d_pair, mem_size_pair);
    CheckError(error);

    error = hipMemcpy(d_pair, pair, mem_size_pair, hipMemcpyHostToDevice);
    CheckError(error);

    //6 point * 2 value(x and y)
    unsigned int mem_size_clipped_vert = npar * 12 * sizeof(float);

    error = hipMalloc((void **) &d_clipped_vert, mem_size_clipped_vert);
	CheckError(error);

	unsigned int mem_size_clipped_n_vert = npar * sizeof(int);
	error = hipMalloc((void **) &d_clipped_n_vert, mem_size_clipped_n_vert);
	CheckError(error);

    setStateInstr();

}

struct polygon
{
	point p[6];
};

__global__ void clip_kernel(triangle *t_s, triangle *t_c, int2 *pair, int npair, polygon *clipped, int *clipped_n)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= npair)
		return;

	triangle *t_s1 = &t_s[pair[idx].x];
	triangle *t_c1 = &t_c[pair[idx].y];

    trgl ts, tc;
    for(int i = 0; i < 3; i++)
    {
        ts.p[i].x = t_s1->p[i].x;
        ts.p[i].y = t_s1->p[i].y;
        tc.p[i].x = t_c1->p[i].x;
        tc.p[i].y = t_c1->p[i].y;
    }

	pt clipped_array[6];
	int clipped_cnt = 0;
	clip(ts, tc, clipped_array, clipped_cnt);
	
	for(int i = 0; i < clipped_cnt; i++)
	{
		clipped[idx].p[i].x = clipped_array[i].x;
		clipped[idx].p[i].y = clipped_array[i].y;
	}
	clipped_n[idx] = clipped_cnt;
}


