#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
#include "stdlib.h"
#include "stdio.h"
#include "clip.h"
#include <iostream>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#define EPS 0.00001
#define N_STATE 11
#define N_INSTR 14

struct polygon
{
	float2 p[6];
};

struct instructSet
{
    bool doIns[N_INSTR];
};

instructSet _stateSet[11];


inline void CheckError(hipError_t error)
{
	if (error != hipSuccess)
	{
		printf("returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}
}

float *d_trgl_s;
float *d_trgl_c;
int2 *d_pair;
polygon *d_clipped_vert;
int *d_clipped_n_vert;
int _npair;
instructSet *d_state;
unsigned int mem_size_clipped_vert;
unsigned int mem_size_clipped_n_vert;

//#if NVCC_ON 
//__constant__ instructSet STATE_SET[N_STATE];
//#endif

//#if NVCC_ON
//__host__
//#endif
void setStateInstr()
{
    for(int s = 0; s < N_STATE; s++)
        for(int i = 0; i < N_INSTR; i++)
            _stateSet[s].doIns[i] = false;

    _stateSet[0].doIns[1] = true;

    _stateSet[1].doIns[0] = true;
    _stateSet[1].doIns[4] = true;

    _stateSet[2].doIns[1] = true;
    _stateSet[2].doIns[5] = true;

    _stateSet[3].doIns[0] = true;
    _stateSet[3].doIns[4] = true;
    _stateSet[3].doIns[6] = true;

    _stateSet[4].doIns[1] = true;
    _stateSet[4].doIns[5] = true;
    _stateSet[4].doIns[7] = true;

    _stateSet[5].doIns[4] = true;
    _stateSet[5].doIns[6] = true;
    _stateSet[5].doIns[8] = true;

    _stateSet[6].doIns[5] = true;
    _stateSet[6].doIns[7] = true;
    _stateSet[6].doIns[9] = true;

    _stateSet[7].doIns[0] = true;
    _stateSet[7].doIns[12] = true;
    _stateSet[7].doIns[2] = true;
    _stateSet[7].doIns[13] = true;
    _stateSet[7].doIns[4] = true;
    _stateSet[7].doIns[6] = true;

    _stateSet[8].doIns[1] = true;
    _stateSet[8].doIns[12] = true;
    _stateSet[8].doIns[3] = true;
    _stateSet[8].doIns[13] = true;
    _stateSet[8].doIns[5] = true;
    _stateSet[8].doIns[7] = true;

    _stateSet[9].doIns[1] = true;
    _stateSet[9].doIns[5] = true;
    _stateSet[9].doIns[10] = true;
    _stateSet[9].doIns[11] = true;

    _stateSet[10].doIns[1] = true;
    _stateSet[10].doIns[3] = true;
    _stateSet[10].doIns[5] = true;

    //hipMemcpyToSymbol(HIP_SYMBOL(STATE_SET),
    //                   &stateSet,
    //                   14 * 11 *sizeof(bool),
    //                   0,
    //                   hipMemcpyHostToDevice);
}




#if NVCC_ON
__host__ __device__
#endif
//touching boundary is also intersect
inline bool BIntersectIncludeBoundary(pt p1, pt p2, pt q1, pt q2)
{
  float  tp, tq, par;

  par = (float) ((p2.x - p1.x)*(q2.y - q1.y) -
                 (p2.y - p1.y)*(q2.x - q1.x));

  if (!par) return 0;                               /* parallel lines */

  tp = ((q1.x - p1.x)*(q2.y - q1.y) - (q1.y - p1.y)*(q2.x - q1.x))/par;
  if(tp<0 || tp>1 )
      return 0;

  tq = ((p2.y - p1.y)*(q1.x - p1.x) - (p2.x - p1.x)*(q1.y - p1.y))/par;
  //touching the boundary is not inside
  if(tq<0 || tq>1)
      return 0;

  return 1;
}



#if NVCC_ON
__host__ __device__
#endif
inline bool BIntersect(pt p1, pt p2, pt q1, pt q2)
{
  float  tp, tq, par;

  par = (float) ((p2.x - p1.x)*(q2.y - q1.y) -
                 (p2.y - p1.y)*(q2.x - q1.x));

  if (!par) return 0;                               /* parallel lines */
  tp = ((q1.x - p1.x)*(q2.y - q1.y) - (q1.y - p1.y)*(q2.x - q1.x))/par;
  tq = ((p2.y - p1.y)*(q1.x - p1.x) - (p2.x - p1.x)*(q1.y - p1.y))/par;

  //touching the boundary is not inside
  if(tp<=0 || tp>=1 || tq<=0 || tq>=1) return 0;

  return 1;
}

#if NVCC_ON
__host__ __device__
#endif
inline point diffPt(pt p1, pt p2)
{
	point p(p1.x - p2.x, p1.y - p2.y);
	return p;
}

#if NVCC_ON
__host__ __device__
#endif
inline float dot(point p1, point p2)
{
	return p1.x * p2.x + p1.y * p2.y;
}

#if NVCC_ON
__host__ __device__
#endif
inline bool testInside(pt p, trgl t)
{
	// Compute vectors        
	point v0 = diffPt(t.p[2], t.p[0]);//C - A
	point v1 = diffPt(t.p[1], t.p[0]);// B - A
	point v2 =  diffPt(p, t.p[0]); //P - A

	// Compute dot products
	float dot00 = dot(v0, v0);
	float dot01 = dot(v0, v1);
	float dot02 = dot(v0, v2);
	float dot11 = dot(v1, v1);
	float dot12 = dot(v1, v2);

	// Compute barycentric coordinates
	float invDenom = 1 / (dot00 * dot11 - dot01 * dot01);
	float u = (dot11 * dot02 - dot01 * dot12) * invDenom;
	float v = (dot00 * dot12 - dot01 * dot02) * invDenom;

	// Check if point is in triangle
	return (u >= 0) && (v >= 0) && (u + v < 1);
}


#if NVCC_ON
__host__ __device__
#endif
inline void Intersect(pt p1, pt p2, pt q1, pt q2,
        pt &pi, pt &qi)
{
    float tp, tq, par;

    par = (float) ((p2.x - p1.x)*(q2.y - q1.y) -
                   (p2.y - p1.y)*(q2.x - q1.x));

    if (!par)
        return;                               /* parallel lines */

    tp = ((q1.x - p1.x)*(q2.y - q1.y) - (q1.y - p1.y)*(q2.x - q1.x))/par;
    tq = ((p2.y - p1.y)*(q1.x - p1.x) - (p2.x - p1.x)*(q1.y - p1.y))/par;

    if(tp<0 || tp>1 || tq<0 || tq>1)
        return;

//    pi.in = true;
//    qi.in = true;
    pi.x = p1.x + tp*(p2.x - p1.x);
    pi.y = p1.y + tp*(p2.y - p1.y);
    qi.x = pi.x;
    qi.y = pi.y;

    //this can be replaced with tp and tq with care
    pi.loc = tp;// dist(p1.x, p1.y, x, y) / dist(p1.x, p1.y, p2.x, p2.y);
    qi.loc = tq;// dist(q1.x, q1.y, x, y) / dist(q1.x, q1.y, q2.x, q2.y);
}




#if NVCC_ON
__host__ __device__
#endif
inline void AddIntersection(trgl ts, trgl tc, pt *clipped_array, int &clipped_cnt)
{
    for(int ic = 0; ic < 3; ic++)
    {
        for(int is = 0; is < 3; is++)
        {
            pt insect_s, insect_c;
            Intersect(tc.p[ic], tc.p[(ic+1)%3], ts.p[is], ts.p[(is+1)%3 ],
                    insect_c, insect_s);

            if(insect_c.loc >= 0)
            {
                insect_c.loc += ic;
                if(clipped_cnt > 0)
                {
                    if(insect_c.loc > clipped_array[clipped_cnt - 1].loc)
                        clipped_array[clipped_cnt++] = insect_c;
                    else if(insect_c.loc < clipped_array[clipped_cnt - 1].loc)
                    {
                        clipped_array[clipped_cnt] = clipped_array[clipped_cnt - 1];
                        clipped_array[clipped_cnt - 1] = insect_c;
                        clipped_cnt++;
                    }
                    //else :insect_c.loc == clipped_vert[isect_cnt - 1].loc
                    //don't add anything
                }
                else
                {
                    clipped_array[0] = insect_c;
                    clipped_cnt++;
                }
            }
        }
    }
}

//have to use __host__ __device__ here, could not recognize template???
#if NVCC_ON
__host__ __device__
#endif
inline void myswap(pt &a, pt &b)
{
	pt tmp = a;
	a = b;
	b = tmp;
}

#if NVCC_ON
__host__
#endif
inline void printTrgl(trgl t)
{
	cout<<"("<<t.p[0].x << ","<< t.p[1].x << "," << t.p[2].x << "," << t.p[0].x<<endl;
	cout<<"("<<t.p[0].y << ","<< t.p[1].y << "," << t.p[2].y << "," << t.p[0].y<<endl;
}

__host__ void GetResultToHost()
{
	hipError_t error;
	
	float *h_clipped_vert = (float*)malloc(mem_size_clipped_vert);
	error = hipMemcpy(h_clipped_vert, d_clipped_vert, mem_size_clipped_vert, hipMemcpyDeviceToHost);
	CheckError(error);

	int *h_clipped_n_vert = (int*)malloc(mem_size_clipped_n_vert);
	error = hipMemcpy(h_clipped_n_vert, d_clipped_n_vert, mem_size_clipped_n_vert, hipMemcpyDeviceToHost);
	CheckError(error);
}


#if NVCC_ON
__host__ __device__
#endif
void clip(trgl ts, trgl tc, pt clipped_array[6], int &clipped_cnt, instructSet *stateInstr)
{
	//mark inside or outside for the triangle vertices
	//and count the number of inside vertices
	int cnt_in_s = 0, cnt_in_c = 0;
	for(int i = 0; i < 3; i++)
	{
		if(tc.p[i].loc = testInside(tc.p[i], ts))
		   cnt_in_c++;

		if(ts.p[i].loc = testInside(ts.p[i], tc))
			cnt_in_s++;
	}

	//make the "in" vertices in the front of the array
	int a[3] = {0, 1, 0};
	for(int i = 0; i < 3; i++)
	{
		int idx = a[i];
		if(!tc.p[idx].loc && tc.p[idx + 1].loc)
			myswap(tc.p[idx], tc.p[idx + 1]);
		if(!ts.p[idx].loc && ts.p[idx + 1].loc)
			myswap(ts.p[idx], ts.p[idx + 1]);
	}

	bool test;
	if(1 == cnt_in_c && 1 == cnt_in_s)
		test = BIntersectIncludeBoundary(ts.p[1], ts.p[2], tc.p[0], tc.p[1]);

	int state = -1;
	if(0 == cnt_in_c && 0 == cnt_in_s)
		state = 0;
	else if(0 == cnt_in_c && 1 == cnt_in_s)
		state = 1;
	else if(1 == cnt_in_c && 0 == cnt_in_s)
		state = 2;
	else if(0 == cnt_in_c && 2 == cnt_in_s)
		state = 3;
	else if(2 == cnt_in_c && 0 == cnt_in_s)
		state = 4;
	else if(0 == cnt_in_c && 3 == cnt_in_s)
		state = 5;
	else if(3 == cnt_in_c && 0 == cnt_in_s)
		state = 6;
	else if(1 == cnt_in_c && 2 == cnt_in_s)
		state = 7;
	else if(2 == cnt_in_c && 1 == cnt_in_s)
		state = 8;
	else if(1 == cnt_in_c && 1 == cnt_in_s && !test)
		state = 9;
	else// if(1 == cnt_in_c && 1 == cnt_in_s && !test1) and (1 == cnt_in_c && 1 == cnt_in_s && test1 && test2)
		state = 10;
	//+cs

    instructSet is = stateInstr[state];
	if(is.doIns[0])//+sc
		AddIntersection(tc, ts, clipped_array, clipped_cnt);
	int tmp = clipped_cnt;
	if(is.doIns[1])//+cs
		AddIntersection(ts, tc, clipped_array, clipped_cnt);
	if(is.doIns[12])
		clipped_array[clipped_cnt] = clipped_array[clipped_cnt - 1];
	if(is.doIns[2])//+c0-
		clipped_array[clipped_cnt - 1] = tc.p[0];
	if(is.doIns[3])//+s0-
		clipped_array[clipped_cnt - 1] = ts.p[0];
	if(is.doIns[13])
		clipped_cnt++;
	if(is.doIns[4])//+s0
		clipped_array[clipped_cnt++] = ts.p[0];
	if(is.doIns[5])//+c0
		clipped_array[clipped_cnt++] = tc.p[0];
	if(is.doIns[6])//+s1
		clipped_array[clipped_cnt++] = ts.p[1];
	if(is.doIns[7])//+c1
		clipped_array[clipped_cnt++] = tc.p[1];
	if(is.doIns[8])//+s2
		clipped_array[clipped_cnt++] = ts.p[2];
	if(is.doIns[9])//+c2
		clipped_array[clipped_cnt++] = tc.p[2];
	if(is.doIns[10])//+r0
		clipped_array[clipped_cnt++] = clipped_array[0];
	if(is.doIns[11])//+r0_s0
		clipped_array[0] = ts.p[0];
}


__global__ void clip_kernel(triangle *t_s, triangle *t_c, int2 *pair, int npair, polygon *clipped, int *clipped_n, instructSet *d_state)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= npair)
		return;

	triangle *t_s1 = &t_s[pair[idx].x];
	triangle *t_c1 = &t_c[pair[idx].y];

    trgl ts, tc;
    for(int i = 0; i < 3; i++)
    {
        ts.p[i].x = t_s1->p[i].x;
        ts.p[i].y = t_s1->p[i].y;
        tc.p[i].x = t_c1->p[i].x;
        tc.p[i].y = t_c1->p[i].y;
    }

	pt clipped_array[6];
	int clipped_cnt = 0;
	clip(ts, tc, clipped_array, clipped_cnt, d_state);
	
	for(int i = 0; i < clipped_cnt; i++)
	{
		clipped[idx].p[i].x = clipped_array[i].x;
		clipped[idx].p[i].y = clipped_array[i].y;
	}
	clipped_n[idx] = clipped_cnt;
}


__host__
vector<point> clip_serial(triangle t_s, triangle t_c)
{
    vector<point> clipped;
    trgl ts, tc;
    int i = 0;
    for(int i = 0; i < 3; i++)
    {
        ts.p[i].x = t_s.p[i].x;
        ts.p[i].y = t_s.p[i].y;
        tc.p[i].x = t_c.p[i].x;
        tc.p[i].y = t_c.p[i].y;
    }
	pt clipped_array[6];
	int clipped_cnt = 0;
	clip(ts, tc, clipped_array, clipped_cnt, _stateSet);

    for(int i = 0; i < clipped_cnt; i++)
    {
        point p(clipped_array[i].x, clipped_array[i].y);
        clipped.push_back(p);
    }
    return clipped;
}

__host__ void finishCUDA()
{
	hipFree(d_clipped_n_vert);
	hipFree(d_clipped_vert);
	hipFree(d_trgl_s);
	hipFree(d_trgl_c);
	hipFree(d_pair);
	hipFree(d_state);
}

__host__ void initCUDA()
{
	int devID = 0;

	hipSetDevice(devID);

	hipError_t error;
	hipDeviceProp_t deviceProp;
	error = hipGetDevice(&devID);

	if (error != hipSuccess)
	{
		printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
	}

	error = hipGetDeviceProperties(&deviceProp, devID);

	if (deviceProp.computeMode == hipComputeModeProhibited)
	{
		fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
		exit(EXIT_SUCCESS);
	}

	if (error != hipSuccess)
	{
		printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
	}
	else
	{
		printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
	}
}

__host__ void loadDataToDevice(float* trgl_s, float* trgl_c, int ntrgl, int *pair, int npair)
{


    hipError_t error;
    unsigned int mem_size = ntrgl * 6 * sizeof(float);//3 vertices, each vertex has x and y(2 float)

    error = hipMalloc((void **) &d_trgl_s, mem_size);
    CheckError(error);

    error = hipMalloc((void **) &d_trgl_c, mem_size);
    CheckError(error);

    error = hipMemcpy(d_trgl_s, trgl_s, mem_size, hipMemcpyHostToDevice);
    CheckError(error);

    error = hipMemcpy(d_trgl_c, trgl_c, mem_size, hipMemcpyHostToDevice);
    CheckError(error);

    unsigned int mem_size_pair = npair * 2 * sizeof(int);

    error = hipMalloc((void **) &d_pair, mem_size_pair);
    CheckError(error);

    error = hipMemcpy(d_pair, pair, mem_size_pair, hipMemcpyHostToDevice);
    CheckError(error);

    //6 point * 2 value(x and y)
    mem_size_clipped_vert = npair * sizeof(polygon);

    error = hipMalloc((void **) &d_clipped_vert, mem_size_clipped_vert);
	CheckError(error);

	mem_size_clipped_n_vert = npair * sizeof(int);
	error = hipMalloc((void **) &d_clipped_n_vert, mem_size_clipped_n_vert);
	CheckError(error);

	//!!!!!!!!!!!!!!!!!!!!
	//assign space for stateSet and copy to device memory
	unsigned int mem_size_state = N_INSTR * N_STATE * sizeof(bool);
	error = hipMalloc((void **) &d_state, mem_size_state);
	error = hipMemcpy(d_state, _stateSet, mem_size_state, hipMemcpyHostToDevice);
    CheckError(error);


	_npair = npair;

}


__global__ void gen_cells_kernel(vtkIdType* cellArray, int N, int* preSum, int* nVert)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= N)
		return;
	
	int begin = idx + preSum[idx];
	int num = nVert[idx];
	int input = preSum[idx];
	cellArray[begin++] = num;
	for(int i = 0; i < num; i++)
		cellArray[begin++] = input++;
}

__global__ void gen_points_kernel(float3 *points, polygon *clipped_vert, int *preSum, int *nVert, int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= N)
		return;

	int num = nVert[idx];
	int begin = preSum[idx];
	for(int i = 0; i < num; i++, begin++)
	{
		points[begin].x = clipped_vert[idx].p[i].x;
		points[begin].y = clipped_vert[idx].p[i].y;
		points[begin].z = 0;
	}
}

__host__
void runKernel(float* &points, vtkIdType* &cells, int &nCells, int &nPts)//triangle *t_s, triangle *t_c, int2 *pair, int npair)//, polygon *clipped, int *clipped_n)
{
	dim3 block(128, 1, 1);
    dim3 grid(ceil((float)_npair / block.x), 1, 1);
	
	clip_kernel<<<grid, block>>>
		((triangle*)d_trgl_s, (triangle*)d_trgl_c, 
		(int2*)d_pair, _npair, 
		d_clipped_vert, d_clipped_n_vert,
		d_state);

	hipError_t error;

	int* d_preSum;
    error = hipMalloc((void **) &d_preSum, mem_size_clipped_n_vert);
    CheckError(error);

	//previous sum for the number of vertices
	thrust::device_ptr<int> d_ptr_clipped_n_vert(d_clipped_n_vert);
	thrust::device_ptr<int> d_ptr_clipped_preSum(d_preSum);
	thrust::exclusive_scan(thrust::device, d_ptr_clipped_n_vert, d_ptr_clipped_n_vert + _npair, d_ptr_clipped_preSum); 

	nPts = d_ptr_clipped_n_vert[_npair - 1] + d_ptr_clipped_preSum[_npair - 1];
	
	///////////points
	float3* d_points;
	unsigned int mem_size_points = nPts * sizeof(float3);
	error = hipMalloc((void **) &d_points, mem_size_points);
	gen_points_kernel<<<grid, block>>>(d_points, d_clipped_vert, d_preSum, d_clipped_n_vert, _npair);

	float3* h_points = (float3*)malloc(mem_size_points);
	error = hipMemcpy(h_points, d_points, mem_size_points, hipMemcpyDeviceToHost);

	//////cells//////
	thrust::device_ptr<int> d_ptr_clipped_n_vert_end = thrust::remove(thrust::device, d_ptr_clipped_n_vert, d_ptr_clipped_n_vert + _npair, 0);
	nCells = d_ptr_clipped_n_vert_end - d_ptr_clipped_n_vert;

	int* d_preSum_compact;
	unsigned int mem_size_preSum_compact = nCells * sizeof(int);
	thrust::device_ptr<int> d_ptr_clipped_preSum_compact(d_preSum_compact);
	thrust::exclusive_scan(thrust::device, d_ptr_clipped_n_vert, d_ptr_clipped_n_vert + nCells, d_ptr_clipped_preSum_compact);

	//cout<<"d_ptr_preSum_compact:"<<endl;
	//for(int i = 0; i < 10; i++)
	//	cout<<d_ptr_clipped_preSum_compact[i]<<endl;

	int size_cells = nPts + nCells;

	unsigned int mem_size_cells = size_cells * sizeof(vtkIdType);

	hipFree(d_clipped_vert);
	hipFree(d_trgl_s);
	hipFree(d_trgl_c);
	hipFree(d_pair);
	hipFree(d_state);
	
	vtkIdType* d_cells;
    error = hipMalloc((void **) &d_cells, mem_size_cells);

	dim3 block2(128, 1, 1);
    dim3 grid2(ceil((float)size_cells / block2.x), 1, 1);
	


	gen_cells_kernel<<<grid2, block2>>>(d_cells, nCells, d_preSum_compact, d_clipped_n_vert);

	vtkIdType* h_cells = (vtkIdType*)malloc(mem_size_cells);
	error = hipMemcpy(h_cells, d_cells, mem_size_cells, hipMemcpyDeviceToHost);
	hipFree(d_cells);
	hipFree(d_clipped_n_vert);
	hipFree(d_preSum);
	hipFree(d_points);

	points = (float*)h_points;
	cells = h_cells;
}