#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#include "stdlib.h"
#include "stdio.h"
#include "clip.h"
#include <iostream>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include "vtkUnstructuredGrid.h"
#include "vtkUnstructuredGridReader.h"
#include <vtkSmartPointer.h>

//#include <math.h>
//#include <hip/hip_math_constants.h>
//#include <math_functions.h>





#define N_STATE 11
#define N_INSTR 14
#define CUDA_ERROR_CHECK
#define RADIUS 1

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

#define M_PI_180 0.01745329252f
#define M_180_PI 57.29577951f
#define M_PI       3.14159265358979323846
#define M_PI_4     0.785398163397448309616
#define M_PI_2     1.57079632679489661923

#define BIN_STEP_X 0.02		//radian
#define BIN_STEP_Y 0.02		//radian

 
inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECKcou
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif
 
    return;
}

inline __host__ __device__ float3 cross(float3 a, float3 b)
{ 
    return make_float3(a.y*b.z - a.z*b.y, a.z*b.x - a.x*b.z, a.x*b.y - a.y*b.x); 
}

inline __host__ __device__ float2 operator-(float2 a, float2 b)
{
    return make_float2(a.x - b.x, a.y - b.y);
}

inline __host__ __device__ float3 operator-(float3 a, float3 b)
{
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}
//
//inline float rsqrtf(float x)
//{
//    return 1.0f / sqrtf(x);
//}

inline __host__ __device__ float dot(float2 a, float2 b)
{ 
    return a.x * b.x + a.y * b.y;
}


inline __host__ __device__ float2 operator*(float2 a, float b)
{
    return make_float2(a.x * b, a.y * b);
}

inline __host__ __device__ float dot(float3 a, float3 b)
{ 
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

inline __host__ __device__ float3 operator*(float3 a, float b)
{
    return make_float3(a.x * b, a.y * b, a.z * b);
}

inline __host__ __device__ float3 operator/(float3 a, float b)
{
    return make_float3(a.x / b, a.y / b, a.z / b);
}


inline __host__ __device__ float3 normalize(float3 v)
{
    float invLen = rsqrtf(dot(v, v));
    return v * invLen;
}

inline __host__ __device__ float length(float3 v)
{
    return sqrtf(dot(v, v));
}

inline __host__ __device__ float3 operator-(float3 &a)
{
    return make_float3(-a.x, -a.y, -a.z);
}


 
inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
 
    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif
 
    return;
}

struct polygon
{
	float2 p[6];
};

struct instructSet
{
    bool doIns[N_INSTR];
};

instructSet _stateSet[11];


inline void CheckError(hipError_t error)
{
	if (error != hipSuccess)
	{
		printf("returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}
}

float *d_trgl_s;
float *d_trgl_c;
int2 *d_pair;
polygon *d_clipped_vert;
int *d_clipped_n_vert;
int _npair;
instructSet *d_state;
unsigned int mem_size_clipped_vert;
unsigned int mem_size_clipped_n_vert;

//#if NVCC_ON 
//__constant__ instructSet STATE_SET[N_STATE];
//#endif


void setStateInstr()
{
    for(int s = 0; s < N_STATE; s++)
        for(int i = 0; i < N_INSTR; i++)
            _stateSet[s].doIns[i] = false;

    _stateSet[0].doIns[1] = true;

    _stateSet[1].doIns[0] = true;
    _stateSet[1].doIns[4] = true;

    _stateSet[2].doIns[1] = true;
    _stateSet[2].doIns[5] = true;

    _stateSet[3].doIns[0] = true;
    _stateSet[3].doIns[4] = true;
    _stateSet[3].doIns[6] = true;

    _stateSet[4].doIns[1] = true;
    _stateSet[4].doIns[5] = true;
    _stateSet[4].doIns[7] = true;

    _stateSet[5].doIns[4] = true;
    _stateSet[5].doIns[6] = true;
    _stateSet[5].doIns[8] = true;

    _stateSet[6].doIns[5] = true;
    _stateSet[6].doIns[7] = true;
    _stateSet[6].doIns[9] = true;

    _stateSet[7].doIns[0] = true;
    _stateSet[7].doIns[12] = true;
    _stateSet[7].doIns[2] = true;
    _stateSet[7].doIns[13] = true;
    _stateSet[7].doIns[4] = true;
    _stateSet[7].doIns[6] = true;

    _stateSet[8].doIns[1] = true;
    _stateSet[8].doIns[12] = true;
    _stateSet[8].doIns[3] = true;
    _stateSet[8].doIns[13] = true;
    _stateSet[8].doIns[5] = true;
    _stateSet[8].doIns[7] = true;

    _stateSet[9].doIns[1] = true;
    _stateSet[9].doIns[5] = true;
    _stateSet[9].doIns[10] = true;
    _stateSet[9].doIns[11] = true;

    _stateSet[10].doIns[1] = true;
    _stateSet[10].doIns[3] = true;
    _stateSet[10].doIns[5] = true;

    //hipMemcpyToSymbol(HIP_SYMBOL(STATE_SET),
    //                   &stateSet,
    //                   14 * 11 *sizeof(bool),
    //                   0,
    //                   hipMemcpyHostToDevice);
}




#if NVCC_ON
__host__ __device__
#endif
//touching boundary is also intersect
inline bool BIntersectIncludeBoundary(float2 p1, float2 p2, float2 q1, float2 q2)
{
  float  tp, tq, par;

  par = (float) ((p2.x - p1.x)*(q2.y - q1.y) -
                 (p2.y - p1.y)*(q2.x - q1.x));

  if (!par) return 0;                               /* parallel lines */

  tp = ((q1.x - p1.x)*(q2.y - q1.y) - (q1.y - p1.y)*(q2.x - q1.x))/par;
  //shouldn't use EPS for 0 here, otherwise the generated triangle has many holes
  if(tp< - EPS || tp> (1 + EPS) )
      return 0;

  tq = ((p2.y - p1.y)*(q1.x - p1.x) - (p2.x - p1.x)*(q1.y - p1.y))/par;
  //touching the boundary is not inside
  if(tq< - EPS || tq> (1 + EPS))
      return 0;

  return 1;
}

#if NVCC_ON
__host__ __device__
#endif
inline bool on_arc(float3 p1, float3 p2, float3 q)
{
	if(length(p1 - q) < EPS || length(p2 - q) < EPS)
		return true;
	else
		return acos(dot(p1, q)) + acos(dot(p2, q)) - acos(dot(p2, p1)) < EPS3;
}

/*
#if NVCC_ON
__host__ __device__
#endif
inline bool BIntersect(float3 p1, float3 p2, float3 q1, float3 q2)
{
	float3 n1 = cross(p1, p2);
	float3 n2 = cross(q1, q2);
	n1 = normalize(n1);
	n2 = normalize(n2);
	if(length(n1 - n2) < EPS3)
		return false;
	else
	{
		float3 L = normalize(cross(n1, n2));
		float3 L_opp = -L;
		return ((on_arc(p1, p2, L) && on_arc(q1, q2, L)) ||
			(on_arc(p1, p2, L_opp) && on_arc(q1, q2, L_opp) ));
	}
}
*/

#if NVCC_ON
__host__ __device__
#endif
inline float3 BigArcIntersectPoint(float3 p1, float3 p2, float3 q1, float3 q2)
{
    float A = p1.y*p2.z-p1.z*p2.y;
    float B = p1.z*p2.x-p1.x*p2.z;
    float C1 = p1.x*p2.y-p1.y*p2.x;
    float D = q1.y*q2.z-q1.z*q2.y;
    float E = q1.z*q2.x-q1.x*q2.z;
    float F = q1.x*q2.y-q1.y*q2.x;

    float BF_CE = B*F - C1 * E;
    float AE_BD = A*E - B*D;
    float AF_CD = A*F - C1 * D;
    float BD_AE = B*D - A*E;

    float3 L = make_float3(BF_CE / AE_BD, AF_CD/ BD_AE, 1.0f);
    return normalize(L);
}

#if NVCC_ON
__host__ __device__
#endif
inline bool onSegment(float3 p1, float3 p2, float3 q1, float3 q2, float3 p1_p2, float3 q1_q2, float3 &p)
{

    float3 p1_p = cross(p1, p);
    float3 p_p2 = cross(p, p2);


    float3 q1_p = cross(q1, p);
    float3 p_q2 = cross(p, q2);


    float d1 = dot(p1_p, p_p2);
    float d2 = dot(p_p2, p1_p2);
    float d3 = dot(q1_p, p_q2);
    float d4 = dot(p_q2, q1_q2);

    return (d1 > EPS3) &&  (d2 > EPS3)
            && (d3 > EPS3) && (d4 > EPS3);
}

//
#if NVCC_ON
__host__ __device__
#endif
inline bool IntersectCore(float3 p1, float3 p2, float3 q1, float3 q2, float3 &p)// &p = make_float3(0,0,0))
{
    //http://mathforum.org/library/drmath/view/62205.html
	//http://www.boeing-727.com/Data/fly%20odds/distance.html
	//  P . (P1 x P2) = 0

    float A = p1.y*p2.z-p1.z*p2.y;
    float B = p1.z*p2.x-p1.x*p2.z;
    float C1 = p1.x*p2.y-p1.y*p2.x;
    float D = q1.y*q2.z-q1.z*q2.y;
    float E = q1.z*q2.x-q1.x*q2.z;
    float F = q1.x*q2.y-q1.y*q2.x;

    float BF_CE = B*F - C1 * E;
    float AE_BD = A*E - B*D;
    float AF_CD = A*F - C1 * D;
    float BD_AE = -AE_BD;

    float3 p1_p2 =  make_float3(A, B, C1);
    float3 q1_q2 = make_float3(D, E, F);
	p =  make_float3(-BF_CE  , AF_CD , -AE_BD);
	
    float len = length(p);

	if(len < EPS3)
		return false;

    p = p / len;

	if(onSegment(p1, p2, q1, q2, p1_p2, q1_q2, p))
        return true;
    p =  -p;
    if(onSegment(p1, p2, q1, q2, p1_p2, q1_q2, p))
        return true;
    return false;
}

#if NVCC_ON
__host__ __device__
#endif
inline bool BIntersect(float3 p1, float3 p2, float3 q1, float3 q2)//, float3 &p = make_float3(0,0,0))
{
    float3 p;
    return IntersectCore(p1, p2, q1, q2, p);

}

#if NVCC_ON
__host__ __device__
#endif
inline void Intersect(float3 p1, float3 p2, float3 q1, float3 q2,
    pt3 &pi)
{
    float3 interPt;
    bool bInter = IntersectCore(p1, p2, q1, q2, interPt);
    if(bInter)
    {
        pi.coord = interPt;
        pi.loc = length(p1 - interPt) / length(p2 - p1);
    }
    return;
}
	/*
inline bool BIntersect(float3 p1, float3 p2, float3 q1, float3 q2)
{
    float3 L = BigArcIntersectPoint(p1, p2, q1, q2);
    float3 L_opp = -L;
    return ((on_arc(p1, p2, L) && on_arc(q1, q2, L)) ||
        (on_arc(p1, p2, L_opp) && on_arc(p1, p2, L_opp) ));
}

#if NVCC_ON
__host__ __device__
#endif
inline void Intersect(float3 p1, float3 p2, float3 q1, float3 q2,
    pt3 &pi)
{
    float3 L = BigArcIntersectPoint(p1, p2, q1, q2);

    float3 L_opp = -L;
    if(on_arc(p1, p2, L) && on_arc(q1, q2, L))
    {
        pi.coord = L;
        pi.loc = length(p1 - L) / length(p2 - p1);
    }
    else if(on_arc(p1, p2, L_opp) && on_arc(q1, q2, L_opp))
    {
        pi.coord = L_opp;
        pi.loc = length(p1 - L_opp) / length(p2 - p1);
    }
    return;
}
*/

#if NVCC_ON
__host__ __device__
#endif
  //touching the boundary is not inside
inline bool BIntersect(float2 p1, float2 p2, float2 q1, float2 q2)
{
  float  tp, tq, par;

  par = (float) ((p2.x - p1.x)*(q2.y - q1.y) -
                 (p2.y - p1.y)*(q2.x - q1.x));

  if (!par) return 0;                               /* parallel lines */
  tp = ((q1.x - p1.x)*(q2.y - q1.y) - (q1.y - p1.y)*(q2.x - q1.x))/par;
	if(tp<EPS || tp> (1 - EPS) )
      return 0;

  tq = ((p2.y - p1.y)*(q1.x - p1.x) - (p2.x - p1.x)*(q1.y - p1.y))/par;

    if(tq<EPS || tq>(1 - EPS))
      return 0;

 // if(tp<=0 || tp>=1 || tq<=0 || tq>=1) return 0;

  return 1;
}

#if NVCC_ON
__host__ __device__
#endif
inline void IntersectIncludeBoundary(float2 p1, float2 p2, float2 q1, float2 q2,
        pt &pi)
{
    float tp, tq, par;

    par = (float) ((p2.x - p1.x)*(q2.y - q1.y) -
                   (p2.y - p1.y)*(q2.x - q1.x));

    if (!par)
        return;                               /* parallel lines */

    tp = ((q1.x - p1.x)*(q2.y - q1.y) - (q1.y - p1.y)*(q2.x - q1.x))/par;
    tq = ((p2.y - p1.y)*(q1.x - p1.x) - (p2.x - p1.x)*(q1.y - p1.y))/par;

    if(tp< - EPS || tp>(1 + EPS) || tq< - EPS || tq> (1 + EPS))
        return;

//    pi.in = true;
//    qi.in = true;
	pi.coord.x = p1.x + tp*(p2.x - p1.x);
    pi.coord.y = p1.y + tp*(p2.y - p1.y);
    //qi.x = pi.x;
    //qi.y = pi.y;

    //this can be replaced with tp and tq with care
    pi.loc = tp;// dist(p1.x, p1.y, x, y) / dist(p1.x, p1.y, p2.x, p2.y);
    //qi.loc = tq;// dist(q1.x, q1.y, x, y) / dist(q1.x, q1.y, q2.x, q2.y);
}

#if NVCC_ON
__host__ __device__
#endif
inline void Intersect(float2 p1, float2 p2, float2 q1, float2 q2,
        pt &pi)
{
    float tp, tq, par;

    par = (float) ((p2.x - p1.x)*(q2.y - q1.y) -
                   (p2.y - p1.y)*(q2.x - q1.x));

    if (!par)
        return;                               /* parallel lines */

    tp = ((q1.x - p1.x)*(q2.y - q1.y) - (q1.y - p1.y)*(q2.x - q1.x))/par;
    tq = ((p2.y - p1.y)*(q1.x - p1.x) - (p2.x - p1.x)*(q1.y - p1.y))/par;

    if(tp<EPS || tp>(1 - EPS) || tq< EPS || tq> (1 - EPS))
        return;

//    pi.in = true;
//    qi.in = true;
	pi.coord.x = p1.x + tp*(p2.x - p1.x);
    pi.coord.y = p1.y + tp*(p2.y - p1.y);
    //qi.x = pi.x;
    //qi.y = pi.y;

    //this can be replaced with tp and tq with care
    pi.loc = tp;// dist(p1.x, p1.y, x, y) / dist(p1.x, p1.y, p2.x, p2.y);
    //qi.loc = tq;// dist(q1.x, q1.y, x, y) / dist(q1.x, q1.y, q2.x, q2.y);
}
/*
#if NVCC_ON
__host__ __device__
#endif
inline void Intersect(float3 p1, float3 p2, float3 q1, float3 q2,
	pt3 &pi)
{
	float3 n1 = cross(p1, p2);
	float3 n2 = cross(q1, q2);
	n1 = normalize(n1);
	n2 = normalize(n2);
	if(length(n1 - n2) >= EPS3)
	{
		float3 L = normalize(cross(n1, n2));
		float3 L_opp = -L;
		if(on_arc(p1, p2, L) && on_arc(q1, q2, L))
		{
			pi.coord = L;
			pi.loc = length(p1 - L) / length(p2 - p1);
		}
		else if(on_arc(p1, p2, L_opp) && on_arc(q1, q2, L_opp))
		{
			pi.coord = L_opp;
			pi.loc = length(p1 - L_opp) / length(p2 - p1);
		}
	}
	return;
}
*/

//
//#if NVCC_ON
//__host__ __device__
//#endif
//inline point diffPt(pt p1, pt p2)
//{
//	point p(p1.x - p2.x, p1.y - p2.y);
//	return p;
//}

//#if NVCC_ON
//__host__ __device__
//#endif
//inline float dot(float2 p1, float2 p2)
//{
//	return p1.x * p2.x + p1.y * p2.y;
//}

//http://forum.beyond3d.com/archive/index.php/t-48658.html
//a, b, c = triangle vertices (in clockwise order)
//x = point on sphere
//
//p1 = dot(x, cross(a, a-c))
//p2 = dot(x, cross(b, b-a))
//p3 = dot(x, cross(c, c-b))

#if NVCC_ON
__host__ __device__
#endif
inline bool testInside(pt p, trgl t)
{
	// Compute vectors        
	float2 v0 = t.p[2].coord - t.p[0].coord;//C - A
	float2 v1 = t.p[1].coord - t.p[0].coord;// B - A
	float2 v2 =  p.coord - t.p[0].coord; //P - A

	// Compute dot products
	float dot00 = dot(v0, v0);
	float dot01 = dot(v0, v1);
	float dot02 = dot(v0, v2);
	float dot11 = dot(v1, v1);
	float dot12 = dot(v1, v2);

	// Compute barycentric coordinates
	float invDenom = 1 / (dot00 * dot11 - dot01 * dot01);
	float u = (dot11 * dot02 - dot01 * dot12) * invDenom;
	float v = (dot00 * dot12 - dot01 * dot02) * invDenom;

	// Check if point is in triangle
	//this EPS has to be very small
	return (u > EPS) && (v > EPS) && (u + v < (1 - EPS));
}


#if NVCC_ON
__host__ __device__
#endif
inline bool testInside(float3 p, trgl3 t)
{  
	/*
	float3 n[3], next, v1, v2;
	for(int i = 0; i < 3; i++)
	{
		next = t.p[(i + 1) % 3].coord;
		v1 = next - p;
		v2 = next - t.p[i].coord;
		n[i] = cross(v1, v2);
		n[i] = normalize(n[i]);
	}

	for(int i = 0; i < 3; i++)
	{
		if(dot(n[i], n[i + 1]) < EPS3)
			return false;
	}
	return true;
	*/
	float3 n[3], e[3];
    float d[3];
    bool b[3];
    for(int i = 0; i < 3; i++)
        n[i] = cross(t.p[(i + 1) % 3].coord ,t.p[(i + 1) % 3].coord - t.p[i].coord);

    for(int i = 0; i < 3; i++)
    {
        d[i] = dot(n[i], p);
        b[i] = d[i] > EPS3;
    }

    if((b[0] && b[1] && b[2]) || (!b[0] && !b[1] && !b[2]))
        return true;

    return false;
}

#if NVCC_ON
__host__ __device__
#endif
inline void AddIntersection(trgl3 ts, trgl3 tc, pt3 *clipped_array, int &clipped_cnt)
{
    for(int ic = 0; ic < 3; ic++)
    {
        for(int is = 0; is < 3; is++)
        {
            pt3 insect_c;
            //Intersect(tc.p[ic], tc.p[(ic+1)%3], ts.p[is], ts.p[(is+1)%3 ],
            //        insect_c, insect_s);
			/*IntersectIncludeBoundary(tc.p[ic], tc.p[(ic+1)%3], ts.p[is], ts.p[(is+1)%3 ],
                    insect_c, insect_s);*/
			Intersect(tc.p[ic].coord, tc.p[(ic+1)%3].coord, ts.p[is].coord, ts.p[(is+1)%3 ].coord, insect_c);

            if(insect_c.loc >= 0)
            {
                insect_c.loc += ic;
                if(clipped_cnt > 0)
                {
					float loc1 = insect_c.loc;
					float loc2 = clipped_array[clipped_cnt - 1].loc;
					//this epsilon could not be too large because loc varies in a small range within [0, 1]
                    if( loc1 - loc2 > EPS)		
                        clipped_array[clipped_cnt++] = insect_c;
                    else if(loc2 - loc1 > EPS)
                    {
                        clipped_array[clipped_cnt] = clipped_array[clipped_cnt - 1];
                        clipped_array[clipped_cnt - 1] = insect_c;
                        clipped_cnt++;
                    }
                    //else :insect_c.loc == clipped_vert[isect_cnt - 1].loc
                    //don't add anything
                }
                else
                {
                    clipped_array[0] = insect_c;
                    clipped_cnt++;
                }
            }
        }
    }
}

#if NVCC_ON
__host__ __device__
#endif
inline void AddIntersection(trgl ts, trgl tc, pt *clipped_array, int &clipped_cnt)
{
    for(int ic = 0; ic < 3; ic++)
    {
        for(int is = 0; is < 3; is++)
        {
            pt insect_c;
            //Intersect(tc.p[ic], tc.p[(ic+1)%3], ts.p[is], ts.p[(is+1)%3 ],
            //        insect_c, insect_s);
			IntersectIncludeBoundary(tc.p[ic].coord, tc.p[(ic+1)%3].coord, ts.p[is].coord, ts.p[(is+1)%3 ].coord,
                    insect_c);

            if(insect_c.loc >= 0)
            {
                insect_c.loc += ic;
                if(clipped_cnt > 0)
                {
					float loc1 = insect_c.loc;
					float loc2 = clipped_array[clipped_cnt - 1].loc;
					//this epsilon could not be too large because loc varies in a small range within [0, 1]
                    if( loc1 - loc2 > EPS)		
                        clipped_array[clipped_cnt++] = insect_c;
                    else if(loc2 - loc1 > EPS)
                    {
                        clipped_array[clipped_cnt] = clipped_array[clipped_cnt - 1];
                        clipped_array[clipped_cnt - 1] = insect_c;
                        clipped_cnt++;
                    }
                    //else :insect_c.loc == clipped_vert[isect_cnt - 1].loc
                    //don't add anything
                }
                else
                {
                    clipped_array[0] = insect_c;
                    clipped_cnt++;
                }
            }
        }
    }
}

//have to use __host__ __device__ here, could not recognize template???
template <typename T>
#if NVCC_ON
__host__ __device__
#endif
inline void myswap(T &a, T &b)
{
	T tmp = a;
	a = b;
	b = tmp;
}

#if NVCC_ON
__host__
#endif
inline void printTrgl(trgl t)
{
	cout<<"("<<t.p[0].coord.x << ","<< t.p[1].coord.x << "," << t.p[2].coord.x << "," << t.p[0].coord.x<<endl;
	cout<<"("<<t.p[0].coord.y << ","<< t.p[1].coord.y << "," << t.p[2].coord.y << "," << t.p[0].coord.y<<endl;
}

inline void printTrgl(triangle t)
{
	cout<<" = ["<<t.p[0].x << ","<< t.p[1].x << "," << t.p[2].x << "," << t.p[0].x<<"];"<<endl;
	cout<<" = ["<<t.p[0].y << ","<< t.p[1].y << "," << t.p[2].y << "," << t.p[0].y<<"];"<<endl;
}

__host__ void GetResultToHost()
{
	hipError_t error;
	
	float *h_clipped_vert = (float*)malloc(mem_size_clipped_vert);
	error = hipMemcpy(h_clipped_vert, d_clipped_vert, mem_size_clipped_vert, hipMemcpyDeviceToHost);
	CudaSafeCall(error);

	int *h_clipped_n_vert = (int*)malloc(mem_size_clipped_n_vert);
	error = hipMemcpy(h_clipped_n_vert, d_clipped_n_vert, mem_size_clipped_n_vert, hipMemcpyDeviceToHost);
	CudaSafeCall(error);
}

__host__ __device__
inline void Geo2Cart(trgl3 &cart, triangle &geo)
{
	float lat;
	float lon;
	for(int i = 0; i < 3; i++)
	{
		lon = geo.p[i].x * M_PI_180;
		lat = geo.p[i].y * M_PI_180;
		cart.p[i].coord.x = RADIUS * cos(lat) * cos(lon);
		cart.p[i].coord.y = RADIUS * cos(lat) * sin(lon);
		cart.p[i].coord.z = RADIUS * sin(lat);
	}
}

__host__ __device__
inline float3 Geo2Cart(float2 geo)
{
	float3 cart;
	float lon = geo.x * M_PI_180;
	float lat = geo.y * M_PI_180;
	cart.x = RADIUS * cos(lat) * cos(lon);
	cart.y = RADIUS * cos(lat) * sin(lon);
	cart.z = RADIUS * sin(lat);
	return cart;
}

__host__ __device__
inline float3 GeoRadian2Cart(float2 geo)
{
	return make_float3(RADIUS * cos(geo.y) * cos(geo.x), RADIUS * cos(geo.y) * sin(geo.x), RADIUS * sin(geo.y));
}

__host__ __device__
inline void Cart2Geo(float2 &geo, float3 &cart)
{
	geo.x = atan2(cart.y, cart.x) * M_180_PI;
	geo.y = asin(cart.z / RADIUS) * M_180_PI;
	if(geo.x < 0)
		geo.x = geo.x + 360;
}

__host__ __device__
inline float3 Cart2Geo(float2 geo)
{
	float3 cart;
	geo.x = atan2(cart.y, cart.x) * M_180_PI;
	geo.y = asin(cart.z / RADIUS) * M_180_PI;
	if(geo.x < 0)
		geo.x = geo.x + 360;
	return cart;
}

__host__ __device__
inline void shrink(pt3 *arr, int size)
{
	for(int i = 1; i < size; i++)
		if(arr[i - 1].loc >= (arr[i].loc ))
			arr[i].loc = -1;
	if(arr[size - 1].loc == (arr[0].loc + 3))
		arr[size - 1].loc = -1;

	int cnt = 1;
	for(int i = 1; i < size; i++)
	{
		if(arr[i].loc != -1)
		{
			arr[cnt++] = arr[i];
		}
	}
}

#if NVCC_ON
__host__ __device__
#endif
void clip3(triangle *t_s1, triangle *t_c1, pt clipped_array_out[6], int &clipped_cnt, instructSet *stateInstr)
{
	trgl3 ts, tc;
	pt3 clipped_array[7];
	Geo2Cart(ts, *t_s1);
	Geo2Cart(tc, *t_c1);
	//mark inside or outside for the triangle vertices
	//and count the number of inside vertices
	int cnt_in_s = 0, cnt_in_c = 0;
	for(int i = 0; i < 3; i++)
	{
		if(tc.p[i].loc = testInside(tc.p[i].coord, ts))
		   cnt_in_c++;

		if(ts.p[i].loc = testInside(ts.p[i].coord, tc))
			cnt_in_s++;
	}

	//make the "in" vertices in the front of the array
	int a[3] = {0, 1, 0};
	for(int i = 0; i < 3; i++)
	{
		int idx = a[i];
		if(tc.p[idx].loc == 0 && tc.p[idx + 1].loc == 1)
			myswap<pt3>(tc.p[idx], tc.p[idx + 1]);
		if(ts.p[idx].loc == 0 && ts.p[idx + 1].loc == 1)
			myswap<pt3>(ts.p[idx], ts.p[idx + 1]);
	}

	bool test;
	if(1 == cnt_in_c && 1 == cnt_in_s)
		//test = BIntersectIncludeBoundary(ts.p[1], ts.p[2], tc.p[0], tc.p[1]);
		test = BIntersect(ts.p[1].coord, ts.p[2].coord, tc.p[0].coord, tc.p[1].coord);

	int state = -1;
	if(0 == cnt_in_c && 0 == cnt_in_s)
		state = 0;
	else if(0 == cnt_in_c && 1 == cnt_in_s)
		state = 1;
	else if(1 == cnt_in_c && 0 == cnt_in_s)
		state = 2;
	else if(0 == cnt_in_c && 2 == cnt_in_s)
		state = 3;
	else if(2 == cnt_in_c && 0 == cnt_in_s)
		state = 4;
	else if(0 == cnt_in_c && 3 == cnt_in_s)
		state = 5;
	else if(3 == cnt_in_c && 0 == cnt_in_s)
		state = 6;
	else if(1 == cnt_in_c && 2 == cnt_in_s)
		state = 7;
	else if(2 == cnt_in_c && 1 == cnt_in_s)
		state = 8;
	else if(1 == cnt_in_c && 1 == cnt_in_s && !test)
		state = 9;
	else// if(1 == cnt_in_c && 1 == cnt_in_s && !test1) and (1 == cnt_in_c && 1 == cnt_in_s && test1 && test2)
		state = 10;
	//+cs

    instructSet is = stateInstr[state];
	if(is.doIns[0])//+sc
		AddIntersection(tc, ts, clipped_array, clipped_cnt);
	if(is.doIns[1])//+cs
		AddIntersection(ts, tc, clipped_array, clipped_cnt);
	if(is.doIns[12])
		clipped_array[clipped_cnt] = clipped_array[clipped_cnt - 1];
	if(is.doIns[2])//+c0-
		clipped_array[clipped_cnt - 1] = tc.p[0];
	if(is.doIns[3])//+s0-
		clipped_array[clipped_cnt - 1] = ts.p[0];
	if(is.doIns[13])
		clipped_cnt++;
	if(is.doIns[4])//+s0
//		if(clipped_array[clipped_cnt - 1].loc < ts.p[0].loc)
		clipped_array[clipped_cnt++] = ts.p[0];
	if(is.doIns[5])//+c0
		clipped_array[clipped_cnt++] = tc.p[0];
	if(is.doIns[6])//+s1
		clipped_array[clipped_cnt++] = ts.p[1];
	if(is.doIns[7])//+c1
		clipped_array[clipped_cnt++] = tc.p[1];
	if(is.doIns[8])//+s2
		clipped_array[clipped_cnt++] = ts.p[2];
	if(is.doIns[9])//+c2
		clipped_array[clipped_cnt++] = tc.p[2];
	if(is.doIns[10])//+r0
		clipped_array[clipped_cnt++] = clipped_array[0];
	if(is.doIns[11])//+r0_s0
		clipped_array[0] = ts.p[0];

//	shrink(clipped_array, clipped_cnt);

	//if number of edge less than 3, then this is not a polygon
	if(clipped_cnt > 0 && clipped_cnt < 3)
	{

	
		//cout<<"state:"<<state<<endl;
		//cout<<"clipped_cnt:"<<clipped_cnt<<endl;
		//cout<<"error:polygon has one or two vertices, impossible case!"<<endl;
		clipped_cnt = 0;
	//	exit(1);
	}
	else if(clipped_cnt > 6)
	{
		//printTrgl(*t_s1);
		//printTrgl(*t_c1);
		//cout<<"state:"<<state<<endl;
		//cout<<"clipped_cnt:"<<clipped_cnt<<endl;
		//exit(1);
		clipped_cnt = 6;
	}

	for(int i = 0; i < clipped_cnt; i++)
	{
		Cart2Geo(clipped_array_out[i].coord, clipped_array[i].coord);
	}
	//clipped_cnt = ts.p[0].x * 1000;//testInside(ts.p[0], tc);
//	clipped_array[0] = ts.p[0];
}


#if NVCC_ON
__host__ __device__
#endif
void clip(triangle *t_s1, triangle *t_c1, pt clipped_array[6], int &clipped_cnt, instructSet *stateInstr)
{
	trgl ts, tc;
    for(int i = 0; i < 3; i++)
    {
		ts.p[i].coord.x = t_s1->p[i].x;
        ts.p[i].coord.y = t_s1->p[i].y;
        tc.p[i].coord.x = t_c1->p[i].x;
        tc.p[i].coord.y = t_c1->p[i].y;
    }
	//mark inside or outside for the triangle vertices
	//and count the number of inside vertices
	int cnt_in_s = 0, cnt_in_c = 0;
	for(int i = 0; i < 3; i++)
	{
		if(tc.p[i].loc = testInside(tc.p[i], ts))
		   cnt_in_c++;

		if(ts.p[i].loc = testInside(ts.p[i], tc))
			cnt_in_s++;
	}

	//make the "in" vertices in the front of the array
	int a[3] = {0, 1, 0};
	for(int i = 0; i < 3; i++)
	{
		int idx = a[i];
		if(tc.p[idx].loc == 0 && tc.p[idx + 1].loc == 1)
			myswap(tc.p[idx], tc.p[idx + 1]);
		if(ts.p[idx].loc == 0 && ts.p[idx + 1].loc == 1)
			myswap(ts.p[idx], ts.p[idx + 1]);
	}

	bool test;
	if(1 == cnt_in_c && 1 == cnt_in_s)
		//test = BIntersectIncludeBoundary(ts.p[1], ts.p[2], tc.p[0], tc.p[1]);
		test = BIntersect(ts.p[1].coord, ts.p[2].coord, tc.p[0].coord, tc.p[1].coord);

	int state = -1;
	if(0 == cnt_in_c && 0 == cnt_in_s)
		state = 0;
	else if(0 == cnt_in_c && 1 == cnt_in_s)
		state = 1;
	else if(1 == cnt_in_c && 0 == cnt_in_s)
		state = 2;
	else if(0 == cnt_in_c && 2 == cnt_in_s)
		state = 3;
	else if(2 == cnt_in_c && 0 == cnt_in_s)
		state = 4;
	else if(0 == cnt_in_c && 3 == cnt_in_s)
		state = 5;
	else if(3 == cnt_in_c && 0 == cnt_in_s)
		state = 6;
	else if(1 == cnt_in_c && 2 == cnt_in_s)
		state = 7;
	else if(2 == cnt_in_c && 1 == cnt_in_s)
		state = 8;
	else if(1 == cnt_in_c && 1 == cnt_in_s && !test)
		state = 9;
	else// if(1 == cnt_in_c && 1 == cnt_in_s && !test1) and (1 == cnt_in_c && 1 == cnt_in_s && test1 && test2)
		state = 10;
	//+cs

    instructSet is = stateInstr[state];
	if(is.doIns[0])//+sc
		AddIntersection(tc, ts, clipped_array, clipped_cnt);
	if(is.doIns[1])//+cs
		AddIntersection(ts, tc, clipped_array, clipped_cnt);
	if(is.doIns[12])
		clipped_array[clipped_cnt] = clipped_array[clipped_cnt - 1];
	if(is.doIns[2])//+c0-
		clipped_array[clipped_cnt - 1] = tc.p[0];
	if(is.doIns[3])//+s0-
		clipped_array[clipped_cnt - 1] = ts.p[0];
	if(is.doIns[13])
		clipped_cnt++;
	if(is.doIns[4])//+s0
		clipped_array[clipped_cnt++] = ts.p[0];
	if(is.doIns[5])//+c0
		clipped_array[clipped_cnt++] = tc.p[0];
	if(is.doIns[6])//+s1
		clipped_array[clipped_cnt++] = ts.p[1];
	if(is.doIns[7])//+c1
		clipped_array[clipped_cnt++] = tc.p[1];
	if(is.doIns[8])//+s2
		clipped_array[clipped_cnt++] = ts.p[2];
	if(is.doIns[9])//+c2
		clipped_array[clipped_cnt++] = tc.p[2];
	if(is.doIns[10])//+r0
		clipped_array[clipped_cnt++] = clipped_array[0];
	if(is.doIns[11])//+r0_s0
		clipped_array[0] = ts.p[0];


	//if number of edge less than 3, then this is not a polygon
	if(clipped_cnt > 0 && clipped_cnt < 3)
	{
	//	printTrgl(ts);
	//	printTrgl(tc);
	//	cout<<"state:"<<state<<endl;
	//	cout<<"clipped_cnt:"<<clipped_cnt<<endl;
		//cout<<"state:"<<state<<endl;
		//cout<<"clipped_cnt:"<<clipped_cnt<<endl;
		//cout<<"error:polygon has one or two vertices, impossible case!"<<endl;
		clipped_cnt = 0;
	//	exit(1);
	}

	//clipped_cnt = ts.p[0].x * 1000;//testInside(ts.p[0], tc);
//	clipped_array[0] = ts.p[0];
}


__global__ void clip_kernel(triangle *t_s, triangle *t_c, int2 *pair, int npair, polygon *clipped, int *clipped_n, instructSet *d_state)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= npair)
		return;

	//triangle *t_s1 = ;
	//triangle *t_c1 = ;



	pt clipped_array[6];
	int clipped_cnt = 0;
	//clip(&t_s[pair[idx].x], &t_c[pair[idx].y], clipped_array, clipped_cnt, d_state);
	clip3(&t_s[pair[idx].x], &t_c[pair[idx].y], clipped_array, clipped_cnt, d_state);
	//if(clipped_cnt > 6)
	//{
	//	clipped_cnt = 7;
	//}
	//
	for(int i = 0; i < clipped_cnt; i++)
	{
		clipped[idx].p[i].x = clipped_array[i].coord.x;
		clipped[idx].p[i].y = clipped_array[i].coord.y;
	}
	//if(clipped_cnt > 6)
	//	asm("trap;");
	clipped_n[idx] = clipped_cnt;
}


__host__
vector<float2> clip_serial(triangle t_s, triangle t_c)
{
    vector<float2> clipped;
    //trgl ts, tc;
    //for(int i = 0; i < 3; i++)
    //{
    //    ts.p[i].x = t_s.p[i].x;
    //    ts.p[i].y = t_s.p[i].y;
    //    tc.p[i].x = t_c.p[i].x;
    //    tc.p[i].y = t_c.p[i].y;
    //}
	pt clipped_array[6];
	int clipped_cnt = 0;
	//clip(&t_s, &t_c, clipped_array, clipped_cnt, _stateSet);
	clip3(&t_s, &t_c, clipped_array, clipped_cnt, _stateSet);

    for(int i = 0; i < clipped_cnt; i++)
    {
		float2 p = make_float2(clipped_array[i].coord.x, clipped_array[i].coord.y);
        clipped.push_back(p);
    }
    return clipped;
}

__host__ void finishCUDA()
{
	hipFree(d_clipped_n_vert);
	hipFree(d_clipped_vert);
	hipFree(d_trgl_s);
	hipFree(d_trgl_c);
	hipFree(d_pair);
	hipFree(d_state);
}

__host__ void initCUDA()
{
	int devID = 0;

	hipSetDevice(devID);

	hipError_t error;
	hipDeviceProp_t deviceProp;
	error = hipGetDevice(&devID);

	if (error != hipSuccess)
	{
		printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
	}

	error = hipGetDeviceProperties(&deviceProp, devID);

	if (deviceProp.computeMode == hipComputeModeProhibited)
	{
		fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
		exit(EXIT_SUCCESS);
	}

	if (error != hipSuccess)
	{
		printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
	}
	else
	{
		printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
	}
}

__host__ void loadDataToDevice(float* trgl_s, float* trgl_c, int ntrgl, int *pair, int npair)
{


    hipError_t error;
    unsigned int mem_size = ntrgl * 6 * sizeof(float);//3 vertices, each vertex has x and y(2 float)

    error = hipMalloc((void **) &d_trgl_s, mem_size);
    CudaSafeCall(error);

    error = hipMalloc((void **) &d_trgl_c, mem_size);
    CudaSafeCall(error);

    error = hipMemcpy(d_trgl_s, trgl_s, mem_size, hipMemcpyHostToDevice);
    CudaSafeCall(error);

    error = hipMemcpy(d_trgl_c, trgl_c, mem_size, hipMemcpyHostToDevice);
    CudaSafeCall(error);

    unsigned int mem_size_pair = npair * 2 * sizeof(int);

    error = hipMalloc((void **) &d_pair, mem_size_pair);
    CudaSafeCall(error);

    error = hipMemcpy(d_pair, pair, mem_size_pair, hipMemcpyHostToDevice);
    CudaSafeCall(error);

    //6 point * 2 value(x and y)
    mem_size_clipped_vert = npair * sizeof(polygon);

    error = hipMalloc((void **) &d_clipped_vert, mem_size_clipped_vert);
	CudaSafeCall(error);

	mem_size_clipped_n_vert = npair * sizeof(int);
	error = hipMalloc((void **) &d_clipped_n_vert, mem_size_clipped_n_vert);
	CudaSafeCall(error);

	//!!!!!!!!!!!!!!!!!!!!
	//assign space for stateSet and copy to device memory
	unsigned int mem_size_state = N_INSTR * N_STATE * sizeof(bool);
	error = hipMalloc((void **) &d_state, mem_size_state);
	error = hipMemcpy(d_state, _stateSet, mem_size_state, hipMemcpyHostToDevice);
    CudaSafeCall(error);


	_npair = npair;

}


__global__ void gen_cells_kernel(vtkIdType* cellArray, int N, int* preSum, int* nVert)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= N)
		return;
	
	int begin = idx + preSum[idx];
	int num = nVert[idx];
	int input = preSum[idx];
	cellArray[begin++] = num;
	for(int i = 0; i < num; i++)
		cellArray[begin++] = input++;
}

__global__ void gen_points_kernel(float3 *points, polygon *clipped_vert, int *preSum, int *nVert, int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= N)
		return;

	int num = nVert[idx];
	int begin = preSum[idx];
	for(int i = 0; i < num; i++, begin++)
	{
		points[begin].x = clipped_vert[idx].p[i].x;
		points[begin].y = clipped_vert[idx].p[i].y;
		points[begin].z = 0;
	}
}

template <typename T>
__host__ void printArray(T *d_array, int size, int num, bool front)
{
	unsigned int mem_size = size * sizeof(T);
	T *h_array;
	h_array = (T*)malloc(mem_size);
	hipError_t error = hipMemcpy(h_array, d_array, mem_size, hipMemcpyDeviceToHost);
    CudaSafeCall(error);
	cout<< "print array:"<<endl;
	for(int i = 0; i < num; i++)
	{
		if(front)
			cout<<h_array[i]<<endl;
		else
			cout<<h_array[size - 1 - i]<<endl;
	}
}

__host__ void printTriangle(triangle* d_trgl, int i)
{
	unsigned int mem_size = sizeof(triangle);
	triangle* h_trgl;
	h_trgl = (triangle*)malloc(mem_size);
	hipError_t error = hipMemcpy(h_trgl, d_trgl + i, mem_size, hipMemcpyDeviceToHost);
    CudaCheckError();
	cout<< "print Triangle:"<<endl;
	for(int j = 0; j < 3; j++)
		cout<<h_trgl->p[j].x<<","<<h_trgl->p[j].y<<endl;
	free(h_trgl);
}



__host__ void printPair(int2 *d_array, int size, int idx)
{
	unsigned int mem_size = size * sizeof(int2);
	int2 *h_array;
	h_array = (int2*)malloc(mem_size);
	hipError_t error = hipMemcpy(h_array, d_array, mem_size, hipMemcpyDeviceToHost);
    CudaSafeCall(error);
	cout<< "print printPair:"<<endl;
	cout<<h_array[idx].x<<","<<h_array[idx].y<<endl;
}

__host__ void printPolygon(polygon *d_array, int size, int idx)
{
	unsigned int mem_size = size * sizeof(polygon);
	polygon *h_array;
	h_array = (polygon*)malloc(mem_size);
	hipError_t error = hipMemcpy(h_array, d_array, mem_size, hipMemcpyDeviceToHost);
    CudaSafeCall(error);
	cout<< "print polygon:"<<endl;
	for(int i = 0; i < 6; i++)
	{
		cout<< h_array[idx].p[i].x << "," << h_array[idx].p[i].y  <<endl;
	}
}



template <typename T>
__host__ void checkArray(T *d_array, int size)
{
	unsigned int mem_size = size * sizeof(T);
	T *h_array;
	h_array = (T*)malloc(mem_size);
	hipError_t error = hipMemcpy(h_array, d_array, mem_size, hipMemcpyDeviceToHost);
    CudaSafeCall(error);
	for(int i = 0; i < size; i++)
	{
		if(h_array[i] > 6)
			cout<<"check:"<<i<<","<<h_array[i]<<endl;
	}
}


__host__
void runKernel(float* &points, vtkIdType* &cells, int &nCells, int &nPts, int nBlock)//triangle *t_s, triangle *t_c, int2 *pair, int npair)//, polygon *clipped, int *clipped_n)
{
	dim3 block(nBlock, 1, 1);
    dim3 grid(ceil((float)_npair / block.x), 1, 1);

	
	//printTriangle((triangle*)d_trgl_s, 16546);
	//printTriangle((triangle*)d_trgl_c, 88008);
	
	clip_kernel<<<grid, block>>>
		((triangle*)d_trgl_s, (triangle*)d_trgl_c, 
		(int2*)d_pair, _npair, 
		d_clipped_vert, d_clipped_n_vert,
		d_state);
	CudaCheckError();



	//printPair(d_pair, _npair, 681046);
	//printPolygon(d_clipped_vert, _npair, 681046);


	//checkArray<int>(d_clipped_n_vert, _npair);

	hipError_t error;

	int* d_preSum;
    error = hipMalloc((void **) &d_preSum, mem_size_clipped_n_vert);
    CudaSafeCall(error);

	//previous sum for the number of vertices
	thrust::device_ptr<int> d_ptr_clipped_n_vert(d_clipped_n_vert);
	//cout<<"num of vert:"<<d_ptr_clipped_n_vert[681046]<<endl;
	thrust::device_ptr<int> d_ptr_clipped_preSum(d_preSum);
	thrust::exclusive_scan(thrust::device, d_ptr_clipped_n_vert, d_ptr_clipped_n_vert + _npair, d_ptr_clipped_preSum); 

	nPts = d_ptr_clipped_n_vert[_npair - 1] + d_ptr_clipped_preSum[_npair - 1];
	//cout<<"nPts:"<<nPts<<endl;
	
	///////////points
	float3* d_points;
	unsigned int mem_size_points = nPts * sizeof(float3);
	error = hipMalloc((void **) &d_points, mem_size_points);
	gen_points_kernel<<<grid, block>>>(d_points, d_clipped_vert, d_preSum, d_clipped_n_vert, _npair);

	float3* h_points = (float3*)malloc(mem_size_points);
	error = hipMemcpy(h_points, d_points, mem_size_points, hipMemcpyDeviceToHost);

	//////cells//////
	thrust::device_ptr<int> d_ptr_clipped_n_vert_end = thrust::remove(thrust::device, d_ptr_clipped_n_vert, d_ptr_clipped_n_vert + _npair, 0);
	nCells = d_ptr_clipped_n_vert_end - d_ptr_clipped_n_vert;

	int* d_preSum_compact;
	unsigned int mem_size_preSum_compact = nCells * sizeof(int);
    hipMalloc((void **) &d_preSum_compact, mem_size_preSum_compact);
	thrust::device_ptr<int> d_ptr_clipped_preSum_compact(d_preSum_compact);
	thrust::exclusive_scan(thrust::device, d_ptr_clipped_n_vert, d_ptr_clipped_n_vert + nCells, d_ptr_clipped_preSum_compact);
	//cout<<"nCells:"<<nCells<<endl;

	
	//cout<<"d_ptr_preSum_compact:"<<endl;
	//for(int i = 0; i < 10; i++)
	//	cout<<d_ptr_clipped_preSum_compact[i]<<endl;

	int size_cells = nPts + nCells;

	unsigned int mem_size_cells = size_cells * sizeof(vtkIdType);


	
	//size_t fr, ttl;
	//hipMemGetInfo(&fr, &ttl);
	//cout<<"fr:"<<fr<<endl;
	//cout<<"ttl:"<<ttl<<endl;

	vtkIdType* d_cells;
    error = hipMalloc((void **) &d_cells, mem_size_cells);
	CudaSafeCall( error );

	dim3 block2(nBlock, 1, 1);
    dim3 grid2(ceil((float)size_cells / block2.x), 1, 1);
	

	cout<<"grid2:"<<grid2.x<<","<<grid2.y<<","<<grid2.z<<endl;
	gen_cells_kernel<<<grid2, block2>>>(d_cells, nCells, d_preSum_compact, d_clipped_n_vert);
	//printArray<vtkIdType>(d_cells, 100, 10, true);
	//printArray<int>(d_preSum_compact, nCells, 10, false);
	//printArray<int>(d_clipped_n_vert, nCells, 10, false);

	vtkIdType* h_cells = (vtkIdType*)malloc(mem_size_cells);
	error = hipMemcpy(h_cells, d_cells, mem_size_cells, hipMemcpyDeviceToHost);
	hipFree(d_cells);
	hipFree(d_clipped_n_vert);
	hipFree(d_preSum);
	hipFree(d_points);
	
	hipFree(d_trgl_s);
	hipFree(d_trgl_c);
	hipFree(d_pair);
	hipFree(d_state);
	hipFree(d_preSum_compact);

	points = (float*)h_points;
	cells = h_cells;
}

struct remove_z
{
	__host__ __device__
	float2 operator() (thrust::tuple<double, double, double> p)
	{
		return make_float2(thrust::get<0>(p), thrust::get<1>(p));
	}
};


struct quad_to_triangles
{
	__host__ __device__
	thrust::tuple<int, int, int, int, int, int> operator() 
		(thrust::tuple<vtkIdType, vtkIdType, vtkIdType, vtkIdType, vtkIdType> cellIdx,
		 thrust::tuple<float, float> coords)
	{
		int i0 = thrust::get<1>(cellIdx);
		int i1 = thrust::get<2>(cellIdx);
		int i2 = thrust::get<3>(cellIdx);
		int i3 = thrust::get<4>(cellIdx);
		return thrust::make_tuple<int, int, int, int, int, int>(i0,i1,i2,i0,i2,i3);
	}
};



typedef thrust::tuple<vtkIdType, vtkIdType, vtkIdType, vtkIdType, vtkIdType> vec5_idtype;
typedef thrust::tuple<float2, float2, float2, float2, float2, float2> trgl2;//coordinates of two triangles
typedef thrust::tuple<float2, float2, float2> trgl1;//coordinates of two triangles

struct assign_triangle_coords
{
	float2* coords;
	assign_triangle_coords(float2* _coords)
	{
		coords = _coords;
	}

	template <typename Tuple>
	__device__ void operator()(Tuple t)
	{
		vec5_idtype cellIdx = thrust::get<0>(t);
		int i0 = thrust::get<1>(cellIdx);
		int i1 = thrust::get<2>(cellIdx);
		int i2 = thrust::get<3>(cellIdx);
		int i3 = thrust::get<4>(cellIdx);

		//compare the distance of diagonal vertices
		//pick the shorter one to devide the quad
		//use radius, instead of degree
		float2 p0 = coords[i0] * M_PI_180;
		float2 p1 = coords[i1] * M_PI_180;
		float2 p2 = coords[i2] * M_PI_180;
		float2 p3 = coords[i3] * M_PI_180;

		float3 p0_cart = GeoRadian2Cart(p0);
		float3 p1_cart = GeoRadian2Cart(p1);
		float3 p2_cart = GeoRadian2Cart(p2);
		float3 p3_cart = GeoRadian2Cart(p3);

		float3 p0_p2 = p2_cart - p0_cart;
		float3 p1_p3 = p3_cart - p1_cart;

		trgl2 twoTrgls;
		if(dot(p0_p2, p0_p2) < dot(p1_p3, p1_p3))
		{
			get<0>(twoTrgls) = p0;
			get<1>(twoTrgls) = p1;
			get<2>(twoTrgls) = p2;
			get<3>(twoTrgls) = p0;
			get<4>(twoTrgls) = p2;
			get<5>(twoTrgls) = p3;
		}
		else
		{
			get<0>(twoTrgls) = p0;
			get<1>(twoTrgls) = p1;
			get<2>(twoTrgls) = p3;
			get<3>(twoTrgls) = p1;
			get<4>(twoTrgls) = p2;
			get<5>(twoTrgls) = p3;
		}
		thrust::get<1>(t) = twoTrgls;
	}
	
};


__device__ inline bool Side0(float radianAngle)
{
	if(radianAngle >= (M_PI_4 * 7) || radianAngle < M_PI_4)
		return true;
	return false;
}

__device__ inline bool Side1(float radianAngle)
{
	if(radianAngle >= (M_PI_4) && radianAngle < (M_PI_4 * 3))
		return true;
	return false;
}

__device__ inline bool Side2(float radianAngle)
{
	if(radianAngle >= (M_PI_4 * 3) && radianAngle < (M_PI_4 * 5))
		return true;
	return false;
}

__device__ inline bool Side3(float radianAngle)
{
	if(radianAngle >= (M_PI_4 * 5) && radianAngle < (M_PI_4 * 7))
		return true;
	return false;
}


__device__ int GetFace(float3 axisAngle)
{
	float2 localCoords;
	if(Side1(axisAngle.y) && Side0(axisAngle.z))
		return 0;
	if(Side3(axisAngle.y) && Side2(axisAngle.z))
		return 1;
	if(Side1(axisAngle.z) && Side0(axisAngle.x))
		return 2;
	if(Side3(axisAngle.z) && Side2(axisAngle.x))
		return 3;
	if(Side1(axisAngle.x) && Side0(axisAngle.y))
		return 4;
	if(Side3(axisAngle.x) && Side2(axisAngle.y))
		return 5;
}


__device__ float2 GetLocalCoords(float3 axisAngle, int face)
{
	float2 localCoords;
	switch(face)
	{
	case 0://Side1(axisAngle.y) && Side0(axisAngle.z)
		localCoords = make_float2(axisAngle.y - M_PI_4, axisAngle.z + M_PI_4);
		break;
	case 1://Side3(axisAngle.y) && Side2(axisAngle.z)
		localCoords = make_float2(axisAngle.y - 5 * M_PI_4, axisAngle.z - 3 * M_PI_4);
		break;
	case 2://Side1(axisAngle.z) && Side0(axisAngle.x)
		localCoords = make_float2(axisAngle.z - M_PI_4, axisAngle.x + M_PI_4);
		break;
	case 3://Side3(axisAngle.z) && Side2(axisAngle.x)
		localCoords = make_float2(axisAngle.z - 5 * M_PI_4, axisAngle.x - 3 * M_PI_4);
		break;
	case 4://Side1(axisAngle.x) && Side0(axisAngle.y)
		localCoords = make_float2(axisAngle.x - M_PI_4, axisAngle.y + M_PI_4);
		break;
	case 5://Side3(axisAngle.x) && Side2(axisAngle.y)
		localCoords = make_float2(axisAngle.x - 5 * M_PI_4, axisAngle.y - 3 * M_PI_4);
		break;
	}
	return localCoords;
}

__device__ int2 GetLocalBin(float2 localCoords)
{
	int2 bin;
	int nBinX = ceil((float)M_PI_2 / BIN_STEP_X);
	int nBinY = ceil((float)M_PI_2 / BIN_STEP_Y);

	if(localCoords.x < 0)
		bin.x = 0;
	else if(localCoords.x > M_PI_2)
		bin.x = nBinX - 1;
	else
		bin.x = localCoords.x / BIN_STEP_X;

	if(localCoords.y < 0)
		bin.y = 0;
	else if(localCoords.y > M_PI_2)
		bin.y = nBinY - 1;
	else
		bin.y = localCoords.y / BIN_STEP_Y;

	return bin;
}

__device__ 

struct functor_getAxisAngle
{
	__host__ __device__
	float3 operator() (float2 p)
	{
		//the angle is in [0, 2*PI]
		float3 axisAngle;
		axisAngle.z = p.x;
		
		float3 cart = GeoRadian2Cart(p);

		axisAngle.x = atan2(cart.z, cart.y);
		if(axisAngle.x < 0)
			axisAngle.x += (2 * M_PI);

		axisAngle.y = atan2(cart.x, cart.z);
		if(axisAngle.y < 0)
			axisAngle.y += (2 * M_PI);

		//axisAngle.z = atan2(cart.y, cart.x);
		//if(axisAngle.z < 0)
		//	axisAngle.z += (2 * M_PI);

		return axisAngle;
	}
};

typedef thrust::tuple<float3, float3, float3> TrglAxisAngle;

__device__ int GetNumBin(TrglAxisAngle t, int face)
{
	int2 bin0 = GetLocalBin(GetLocalCoords(thrust::get<0>(t), face));
	int2 bin1 = GetLocalBin(GetLocalCoords(thrust::get<1>(t), face));
	int2 bin2 = GetLocalBin(GetLocalCoords(thrust::get<2>(t), face));

	int2 min;
	int2 max;

	min.x = min3(bin0.x, bin1.x, bin2.x);
	min.y = min3(bin0.y, bin1.y, bin2.y);
	max.x = max3(bin0.x, bin1.x, bin2.x);
	max.y = max3(bin0.y, bin1.y, bin2.y);

	return (max.x - min.x + 1) * (max.y - min.y + 1);
}

__device__ int getBin(int face, int ix, int iy)
{
	int nBinX = ceil((float)M_PI_2 / BIN_STEP_X);
	int nBinY = ceil((float)M_PI_2 / BIN_STEP_Y);
	return (nBinX * nBinY * face + nBinX * iy + ix);
}

__device__ void GetSearchPair(TrglAxisAngle t, int face, int2* &writeCursor, int trglIdx)
{
	int2 bin0 = GetLocalBin(GetLocalCoords(thrust::get<0>(t), face));
	int2 bin1 = GetLocalBin(GetLocalCoords(thrust::get<1>(t), face));
	int2 bin2 = GetLocalBin(GetLocalCoords(thrust::get<2>(t), face));

	int2 min;
	int2 max;

	min.x = min3(bin0.x, bin1.x, bin2.x);
	min.y = min3(bin0.y, bin1.y, bin2.y);
	max.x = max3(bin0.x, bin1.x, bin2.x);
	max.y = max3(bin0.y, bin1.y, bin2.y);

	for(int iy = min.y; iy <= max.y; iy++)
	{
		for(int ix = min.x; ix <= max.x; ix++)
		{
			//int2(bin index, triangle index)
			*writeCursor = make_int2(getBin(face, ix, iy), trglIdx);//make_int2(ix,iy);//
			writeCursor++;
		}
	}
}

struct functor_getNumBin
{
	 __device__
	int operator() (TrglAxisAngle t)
	{
		int nBin;
		float3 v0 = thrust::get<0>(t);
		float3 v1 = thrust::get<1>(t);
		float3 v2 = thrust::get<2>(t);
		int f0 = GetFace(v0);
		int f1 = GetFace(v1);
		int f2 = GetFace(v2);
	//	return f2;// v0.x * 1000;//(abs(f0 - f1) < EPS);//((f0 != f1));//

		if((abs(f0 - f1) < EPS) && (abs(f1 - f2) < EPS)) //three vertices are all in one face
		{
			nBin = GetNumBin(t, f0);
		}
		else if(abs(f0 - f1) < EPS)	//on two face
		{
			nBin = GetNumBin(t, f1) + GetNumBin(t, f2);
		}
		else if(abs(f1 - f2) < EPS)	//on two face
		{
			nBin = GetNumBin(t, f2) + GetNumBin(t, f0);
		}
		else if(abs(f2 - f0) < EPS)	//on two face
		{
			nBin = GetNumBin(t, f0) + GetNumBin(t, f1);
		}
		else	//on three different faces
		{
			nBin = GetNumBin(t, f0) + GetNumBin(t, f1) + GetNumBin(t, f2);
		}
		return nBin;
	}
};

struct functor_fillSearchStruct
{
	int2* searchStruct;
	functor_fillSearchStruct(int2* _searchStruct)
	{
		searchStruct = _searchStruct;
	}

	__device__ void operator() (thrust::tuple<TrglAxisAngle, int, int> tup)
	{
		TrglAxisAngle t = thrust::get<0>(tup);
		int offset = thrust::get<1>(tup);
		int trglIdx = thrust::get<2>(tup);
		int2* writeCursor = searchStruct + offset;

		float3 v0 = thrust::get<0>(t);
		float3 v1 = thrust::get<1>(t);
		float3 v2 = thrust::get<2>(t);
		int f0 = GetFace(v0);
		int f1 = GetFace(v1);
		int f2 = GetFace(v2);

		if((abs(f0 - f1) < EPS) && (abs(f1 - f2) < EPS)) //three vertices are all in one face
		{
			GetSearchPair(t, f0, writeCursor, trglIdx);
		}
		else if(abs(f0 - f1) < EPS)	//on two face
		{
			GetSearchPair(t, f1, writeCursor, trglIdx);
			GetSearchPair(t, f2, writeCursor, trglIdx);
		}
		else if(abs(f1 - f2) < EPS)	//on two face
		{
			GetSearchPair(t, f2, writeCursor, trglIdx);
			GetSearchPair(t, f0, writeCursor, trglIdx);
		}
		else if(abs(f2 - f0) < EPS)	//on two face
		{
			GetSearchPair(t, f0, writeCursor, trglIdx);
			GetSearchPair(t, f1, writeCursor, trglIdx);
		}
		else	//on three different faces
		{
			GetSearchPair(t, f0, writeCursor, trglIdx);
			GetSearchPair(t, f1, writeCursor, trglIdx);
			GetSearchPair(t, f2, writeCursor, trglIdx);
		}
	}
};

struct BinCmp {
	__host__ __device__
	bool operator()(const int2& v1, const int2& v2) {
		return v1.x < v2.x;
	}
};

void GetPairs(vtkPoints* vtkPts_s, vtkCellArray* vtkCls_s, thrust::device_vector<int2> &d_vec_searchStruct)//, int &numBins)
{
	thrust::tuple<double, double, double>* pointCoords_s = (thrust::tuple<double, double, double>*)vtkPts_s->GetVoidPointer(0);
	int nPoints = vtkPts_s->GetNumberOfPoints();
	clock_t t_1 = clock();
	thrust::device_vector<thrust::tuple<double, double, double>> d_vec_vtkPtsCoords_s
		(pointCoords_s, pointCoords_s + nPoints);
	clock_t t0 = clock();
	unsigned long compute_time = (t0 - t_1) * 1000 / CLOCKS_PER_SEC;
    cout<<"loading VTK point data:"<< (float)compute_time * 0.001 << "sec" << endl;

	thrust::device_vector<float2> d_vec_vtkPts_s(nPoints);
	
	
	
	transform(d_vec_vtkPtsCoords_s.begin(), d_vec_vtkPtsCoords_s.end(), d_vec_vtkPts_s.begin(), remove_z());
	clock_t t1 = clock();
	compute_time = (t1 - t0) * 1000 / CLOCKS_PER_SEC;
    cout<<"remove z coordinate:"<< (float)compute_time * 0.001 << "sec" << endl;
	/**********Cells********/
	int nCells_s = vtkCls_s->GetNumberOfCells();

	vtkIdType* cellIdx_s = vtkCls_s->GetData()->GetPointer(0);
	int sizeCellArray_s = vtkCls_s->GetSize();

	//input: point index of quad
	thrust::device_vector<vtkIdType> d_vec_vtkCls_s(cellIdx_s,
		cellIdx_s + sizeCellArray_s);
	//device pointer to one quad indices
	thrust::device_ptr<vec5_idtype> d_vec_vtkCls_vec5_s = 
		thrust::device_ptr<vec5_idtype>((vec5_idtype*)raw_pointer_cast( &d_vec_vtkCls_s[0]));

	//output: point index of two triangles
	thrust::device_vector<trgl2> trglCoords_s(nCells_s);

	//input: points coordinates(globally access)
	float2* ptsCoords_s = thrust::raw_pointer_cast(d_vec_vtkPts_s.data());


	//computing
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(d_vec_vtkCls_vec5_s, trglCoords_s.begin())), 
		thrust::make_zip_iterator(thrust::make_tuple(d_vec_vtkCls_vec5_s + nCells_s, trglCoords_s.end())), 
		assign_triangle_coords(ptsCoords_s));

	clock_t t2 = clock();
    compute_time = (t2 - t1) * 1000 / CLOCKS_PER_SEC;
    cout<<"time to generate array to put triangle coordinates:"<< (float)compute_time * 0.001 << "sec" << endl;

	//for(int i = 0; i < 4; i++)
	//{
	//	trgl2 t2 = trglCoords_s[i];
	//	cout<< thrust::get<0>(t2).x<<","<<thrust::get<0>(t2).y<<endl;
	//	cout<< thrust::get<1>(t2).x<<","<<thrust::get<1>(t2).y<<endl;
	//	cout<< thrust::get<2>(t2).x<<","<<thrust::get<2>(t2).y<<endl;
	//	cout<< thrust::get<3>(t2).x<<","<<thrust::get<3>(t2).y<<endl;
	//	cout<< thrust::get<4>(t2).x<<","<<thrust::get<4>(t2).y<<endl;
	//	cout<< thrust::get<5>(t2).x<<","<<thrust::get<5>(t2).y<<endl;
	//}

	//input
	thrust::device_ptr<float2> d_ptr_pointGeoCoords_s = 
		thrust::device_ptr<float2>((float2*)raw_pointer_cast(&trglCoords_s[0]));
	//output:
	//each cell has 2 triangle, each triangle has 3 points
	int nVertex = nCells_s * 3 * 2;
	thrust::device_vector<float3> d_vec_pointAxisAngle_s(nVertex);

	//computing
	thrust::transform(d_ptr_pointGeoCoords_s, d_ptr_pointGeoCoords_s + nVertex, 
		d_vec_pointAxisAngle_s.begin(), functor_getAxisAngle());
	clock_t t3 = clock();
    compute_time = (t3 - t2) * 1000 / CLOCKS_PER_SEC;
     cout<<"time to compute axis angle:"<< (float)compute_time * 0.001 << "sec" << endl;
	//cout<<"axis angle:"<<endl;
	//for(int i = 0; i < 4; i++)
	//{
	//	trgl2 t2 = trglCoords_s[i];
	//	cout<< ((float3)d_vec_pointAxisAngle_s.data()[i]).x<<","
	//		<<((float3)d_vec_pointAxisAngle_s.data()[i]).y<<","
	//		<<((float3)d_vec_pointAxisAngle_s.data()[i]).z
	//		<<endl;
	//}
	
	//output: number of Bins for each triangle
	int nTrgl = nCells_s * 2;
	thrust::device_vector<int> d_vec_numBinPerTrgl(nTrgl);
	//input:
	thrust::device_ptr<TrglAxisAngle> d_ptr_trglAxisAngle_s
		((TrglAxisAngle*)raw_pointer_cast(d_vec_pointAxisAngle_s.data())) ;

	//compute the number of bins, each triangle falls in
	thrust::transform(d_ptr_trglAxisAngle_s, d_ptr_trglAxisAngle_s + nTrgl, 
		d_vec_numBinPerTrgl.begin(), functor_getNumBin());

	clock_t t4 = clock();
    compute_time = (t4 - t3) * 1000 / CLOCKS_PER_SEC;
     cout<<"time to compute the number of bins, each triangle falls in:"<< (float)compute_time * 0.001 << "sec" << endl;
	//cout<<"nTrgl:"<<nTrgl<<endl;
	/*cout<<"number of bins:"<<endl;
	for(int i = 2708; i < 2712; i ++)
	{
		cout<<d_vec_numBinPerTrgl[i]<<endl;
	}*/
	//input:
	thrust::device_vector<int> d_vec_searchStructOffset(nTrgl);
	//compute:
	thrust::exclusive_scan(thrust::device, d_vec_numBinPerTrgl.begin(), d_vec_numBinPerTrgl.end(), 
		d_vec_searchStructOffset.begin()); 

		clock_t t5 = clock();
    compute_time = (t5 - t4) * 1000 / CLOCKS_PER_SEC;
     cout<<"time to do scan for offset:"<< (float)compute_time * 0.001 << "sec" << endl;

	//cout<<"offset:"<<endl;
	//for(int i = 0; i < 32; i++)
	//	cout<<d_vec_searchStructOffset[i]<<endl;

	int numBins = d_vec_searchStructOffset.back() + d_vec_numBinPerTrgl.back();
	//cout<<"d_vec_searchStructOffset[nTrgl - 1]:"<<d_vec_searchStructOffset[nTrgl - 1]<<endl;
	//cout<<"d_vec_numBinPerTrgl[nTrgl - 1]:"<<d_vec_numBinPerTrgl[nTrgl - 1]<<endl;
	//cout<<"numBins:"<<numBins<<endl;

	//input::triangle index
	thrust::counting_iterator<int> first(0);
	thrust::counting_iterator<int> last = first + nTrgl;
	//output: search structure int2(bin index, triangle index)
	//thrust::device_vector<int2> d_vec_searchStruct(numBins);
	d_vec_searchStruct.resize(numBins);
	int2* d_raw_ptr_searchStruct = raw_pointer_cast(d_vec_searchStruct.data());
	//compute search structure:
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(d_ptr_trglAxisAngle_s, d_vec_searchStructOffset.begin(),first)),
		thrust::make_zip_iterator(thrust::make_tuple(d_ptr_trglAxisAngle_s + nTrgl, d_vec_searchStructOffset.end(), last)), 
		functor_fillSearchStruct(d_raw_ptr_searchStruct));

		clock_t t6 = clock();
    compute_time = (t6 - t5) * 1000 / CLOCKS_PER_SEC;
     cout<<"time to input bins for each triangle:"<< (float)compute_time * 0.001 << "sec" << endl;

	thrust::device_ptr<int2> d_ptr_searchStruct(d_raw_ptr_searchStruct);


	//for(int i = 3500; i < numBins; i++)
	//{
	//	int2 temp = d_ptr_searchStruct[i];

	//	cout<<temp.x <<","<<temp.y<<endl;
	//	if(temp.x == 0)
	//		exit(1);
	//}

	

	//sort based on bin number
	thrust::sort(d_ptr_searchStruct, d_ptr_searchStruct + numBins, BinCmp());
	//for(int i = 0; i < 500; i++)
	//{
	//	int2 temp = d_ptr_searchStruct[i];
	//	cout<<temp.x <<","<<temp.y<<endl;
	//}
		clock_t t7 = clock();
	compute_time = (t7 - t6) * 1000 / CLOCKS_PER_SEC;
     cout<<"time to sort based on bin number:"<< (float)compute_time * 0.001 << "sec" << endl;

//	return d_vec_searchStruct;
//	searchStruct = d_ptr_searchStruct;
}


__host__ void runCUDA(/*vtkPoints* vtkPts_s, vtkCellArray* vtkCls_s, vtkPoints* vtkPts_c, vtkCellArray* vtkCls_c,*/
	char* filename_subject, char* filename_constraint)
{
	thrust::device_vector<int2> searchStruct_s, searchStruct_c;
	int numBins_s, numBins_c;
	

    //clock_t t0 = clock();
	vtkSmartPointer<vtkUnstructuredGridReader> reader =
      vtkSmartPointer<vtkUnstructuredGridReader>::New();
	vtkSmartPointer<vtkUnstructuredGridReader> reader_c =
      vtkSmartPointer<vtkUnstructuredGridReader>::New();

	//reading subject file
    reader->SetFileName(filename_subject);
    reader->Update();
    vtkUnstructuredGrid* grid_s = reader->GetOutput();
    vtkPoints* points_s = grid_s->GetPoints();
    vtkCellArray* cell_s = grid_s->GetCells();
    //reader->CloseVTKFile();


	//reading constriant file
	reader_c->SetFileName(filename_constraint);
    reader_c->Update(); // Needed because of GetScalarRange
    vtkUnstructuredGrid* grid_c = reader_c->GetOutput();
    vtkPoints* points_c = grid_c->GetPoints();
    vtkCellArray* cell_c = grid_c->GetCells();
   // reader->CloseVTKFile();
	
	
	GetPairs(points_s, cell_s, searchStruct_s);//, numBins_s);
	GetPairs(points_c, cell_c, searchStruct_c);//, numBins_c);
	
	cout<<"numBins_s:"<<searchStruct_s.size()<<endl;
	cout<<"numBins_c:"<<searchStruct_c.size()<<endl;

	
	/*clock_t t1 = clock();
    unsigned long compute_time = (t1 - t0) * 1000 / CLOCKS_PER_SEC;
    cout<<"time to get pair <bin number, triangle number>:"<< (float)compute_time * 0.001 << "sec" << endl;
*/
	/*cout<<"pair_s:"<<endl;
	for(int i = 0; i < 50; i++)
	{
		int2 temp = searchStruct_s[i];
		cout<<temp.x <<","<<temp.y<<endl;
	}

	cout<<"pair_c:"<<endl;
	for(int i = 0; i < 50; i++)
	{
		int2 temp = searchStruct_c[i];
		cout<<temp.x <<","<<temp.y<<endl;
	}*/
}