#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#include "stdlib.h"
#include "stdio.h"
#include "clip.h"
#include <iostream>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#define N_STATE 11
#define N_INSTR 14
#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )
 
inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif
 
    return;
}
 
inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
 
    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif
 
    return;
}

struct polygon
{
	float2 p[6];
};

struct instructSet
{
    bool doIns[N_INSTR];
};

instructSet _stateSet[11];


inline void CheckError(hipError_t error)
{
	if (error != hipSuccess)
	{
		printf("returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}
}

float *d_trgl_s;
float *d_trgl_c;
int2 *d_pair;
polygon *d_clipped_vert;
int *d_clipped_n_vert;
int _npair;
instructSet *d_state;
unsigned int mem_size_clipped_vert;
unsigned int mem_size_clipped_n_vert;

//#if NVCC_ON 
//__constant__ instructSet STATE_SET[N_STATE];
//#endif


void setStateInstr()
{
    for(int s = 0; s < N_STATE; s++)
        for(int i = 0; i < N_INSTR; i++)
            _stateSet[s].doIns[i] = false;

    _stateSet[0].doIns[1] = true;

    _stateSet[1].doIns[0] = true;
    _stateSet[1].doIns[4] = true;

    _stateSet[2].doIns[1] = true;
    _stateSet[2].doIns[5] = true;

    _stateSet[3].doIns[0] = true;
    _stateSet[3].doIns[4] = true;
    _stateSet[3].doIns[6] = true;

    _stateSet[4].doIns[1] = true;
    _stateSet[4].doIns[5] = true;
    _stateSet[4].doIns[7] = true;

    _stateSet[5].doIns[4] = true;
    _stateSet[5].doIns[6] = true;
    _stateSet[5].doIns[8] = true;

    _stateSet[6].doIns[5] = true;
    _stateSet[6].doIns[7] = true;
    _stateSet[6].doIns[9] = true;

    _stateSet[7].doIns[0] = true;
    _stateSet[7].doIns[12] = true;
    _stateSet[7].doIns[2] = true;
    _stateSet[7].doIns[13] = true;
    _stateSet[7].doIns[4] = true;
    _stateSet[7].doIns[6] = true;

    _stateSet[8].doIns[1] = true;
    _stateSet[8].doIns[12] = true;
    _stateSet[8].doIns[3] = true;
    _stateSet[8].doIns[13] = true;
    _stateSet[8].doIns[5] = true;
    _stateSet[8].doIns[7] = true;

    _stateSet[9].doIns[1] = true;
    _stateSet[9].doIns[5] = true;
    _stateSet[9].doIns[10] = true;
    _stateSet[9].doIns[11] = true;

    _stateSet[10].doIns[1] = true;
    _stateSet[10].doIns[3] = true;
    _stateSet[10].doIns[5] = true;

    //hipMemcpyToSymbol(HIP_SYMBOL(STATE_SET),
    //                   &stateSet,
    //                   14 * 11 *sizeof(bool),
    //                   0,
    //                   hipMemcpyHostToDevice);
}




#if NVCC_ON
__host__ __device__
#endif
//touching boundary is also intersect
inline bool BIntersectIncludeBoundary(pt p1, pt p2, pt q1, pt q2)
{
  float  tp, tq, par;

  par = (float) ((p2.x - p1.x)*(q2.y - q1.y) -
                 (p2.y - p1.y)*(q2.x - q1.x));

  if (!par) return 0;                               /* parallel lines */

  tp = ((q1.x - p1.x)*(q2.y - q1.y) - (q1.y - p1.y)*(q2.x - q1.x))/par;
  //shouldn't use EPS for 0 here, otherwise the generated triangle has many holes
  if(tp< - EPS || tp> (1 + EPS) )
      return 0;

  tq = ((p2.y - p1.y)*(q1.x - p1.x) - (p2.x - p1.x)*(q1.y - p1.y))/par;
  //touching the boundary is not inside
  if(tq< - EPS || tq> (1 + EPS))
      return 0;

  return 1;
}



#if NVCC_ON
__host__ __device__
#endif
  //touching the boundary is not inside
inline bool BIntersect(pt p1, pt p2, pt q1, pt q2)
{
  float  tp, tq, par;

  par = (float) ((p2.x - p1.x)*(q2.y - q1.y) -
                 (p2.y - p1.y)*(q2.x - q1.x));

  if (!par) return 0;                               /* parallel lines */
  tp = ((q1.x - p1.x)*(q2.y - q1.y) - (q1.y - p1.y)*(q2.x - q1.x))/par;
	if(tp<EPS || tp> (1 - EPS) )
      return 0;

  tq = ((p2.y - p1.y)*(q1.x - p1.x) - (p2.x - p1.x)*(q1.y - p1.y))/par;

    if(tq<EPS || tq>(1 - EPS))
      return 0;

 // if(tp<=0 || tp>=1 || tq<=0 || tq>=1) return 0;

  return 1;
}

#if NVCC_ON
__host__ __device__
#endif
inline void IntersectIncludeBoundary(pt p1, pt p2, pt q1, pt q2,
        pt &pi, pt &qi)
{
    float tp, tq, par;

    par = (float) ((p2.x - p1.x)*(q2.y - q1.y) -
                   (p2.y - p1.y)*(q2.x - q1.x));

    if (!par)
        return;                               /* parallel lines */

    tp = ((q1.x - p1.x)*(q2.y - q1.y) - (q1.y - p1.y)*(q2.x - q1.x))/par;
    tq = ((p2.y - p1.y)*(q1.x - p1.x) - (p2.x - p1.x)*(q1.y - p1.y))/par;

    if(tp< - EPS || tp>(1 + EPS) || tq< - EPS || tq> (1 + EPS))
        return;

//    pi.in = true;
//    qi.in = true;
    pi.x = p1.x + tp*(p2.x - p1.x);
    pi.y = p1.y + tp*(p2.y - p1.y);
    qi.x = pi.x;
    qi.y = pi.y;

    //this can be replaced with tp and tq with care
    pi.loc = tp;// dist(p1.x, p1.y, x, y) / dist(p1.x, p1.y, p2.x, p2.y);
    qi.loc = tq;// dist(q1.x, q1.y, x, y) / dist(q1.x, q1.y, q2.x, q2.y);
}

#if NVCC_ON
__host__ __device__
#endif
inline void Intersect(pt p1, pt p2, pt q1, pt q2,
        pt &pi, pt &qi)
{
    float tp, tq, par;

    par = (float) ((p2.x - p1.x)*(q2.y - q1.y) -
                   (p2.y - p1.y)*(q2.x - q1.x));

    if (!par)
        return;                               /* parallel lines */

    tp = ((q1.x - p1.x)*(q2.y - q1.y) - (q1.y - p1.y)*(q2.x - q1.x))/par;
    tq = ((p2.y - p1.y)*(q1.x - p1.x) - (p2.x - p1.x)*(q1.y - p1.y))/par;

    if(tp<EPS || tp>(1 - EPS) || tq< EPS || tq> (1 - EPS))
        return;

//    pi.in = true;
//    qi.in = true;
    pi.x = p1.x + tp*(p2.x - p1.x);
    pi.y = p1.y + tp*(p2.y - p1.y);
    qi.x = pi.x;
    qi.y = pi.y;

    //this can be replaced with tp and tq with care
    pi.loc = tp;// dist(p1.x, p1.y, x, y) / dist(p1.x, p1.y, p2.x, p2.y);
    qi.loc = tq;// dist(q1.x, q1.y, x, y) / dist(q1.x, q1.y, q2.x, q2.y);
}

#if NVCC_ON
__host__ __device__
#endif
inline point diffPt(pt p1, pt p2)
{
	point p(p1.x - p2.x, p1.y - p2.y);
	return p;
}

#if NVCC_ON
__host__ __device__
#endif
inline float dot(point p1, point p2)
{
	return p1.x * p2.x + p1.y * p2.y;
}

#if NVCC_ON
__host__ __device__
#endif
inline bool testInside(pt p, trgl t)
{
	// Compute vectors        
	point v0 = diffPt(t.p[2], t.p[0]);//C - A
	point v1 = diffPt(t.p[1], t.p[0]);// B - A
	point v2 =  diffPt(p, t.p[0]); //P - A

	// Compute dot products
	double dot00 = dot(v0, v0);
	double dot01 = dot(v0, v1);
	double dot02 = dot(v0, v2);
	double dot11 = dot(v1, v1);
	double dot12 = dot(v1, v2);

	// Compute barycentric coordinates
	double invDenom = 1 / (dot00 * dot11 - dot01 * dot01);
	float u = (dot11 * dot02 - dot01 * dot12) * invDenom;
	float v = (dot00 * dot12 - dot01 * dot02) * invDenom;

	// Check if point is in triangle
	//this EPS has to be very small
	return (u > EPS) && (v > EPS) && (u + v < (1 - EPS));
}







#if NVCC_ON
__host__ __device__
#endif
inline void AddIntersection(trgl ts, trgl tc, pt *clipped_array, int &clipped_cnt)
{
    for(int ic = 0; ic < 3; ic++)
    {
        for(int is = 0; is < 3; is++)
        {
            pt insect_s, insect_c;
            //Intersect(tc.p[ic], tc.p[(ic+1)%3], ts.p[is], ts.p[(is+1)%3 ],
            //        insect_c, insect_s);
			IntersectIncludeBoundary(tc.p[ic], tc.p[(ic+1)%3], ts.p[is], ts.p[(is+1)%3 ],
                    insect_c, insect_s);

            if(insect_c.loc >= 0)
            {
                insect_c.loc += ic;
                if(clipped_cnt > 0)
                {
					float loc1 = insect_c.loc;
					float loc2 = clipped_array[clipped_cnt - 1].loc;
					//this epsilon could not be too large because loc varies in a small range within [0, 1]
                    if( loc1 - loc2 > EPS2)		
                        clipped_array[clipped_cnt++] = insect_c;
                    else if(loc2 - loc1 > EPS2)
                    {
                        clipped_array[clipped_cnt] = clipped_array[clipped_cnt - 1];
                        clipped_array[clipped_cnt - 1] = insect_c;
                        clipped_cnt++;
                    }
                    //else :insect_c.loc == clipped_vert[isect_cnt - 1].loc
                    //don't add anything
                }
                else
                {
                    clipped_array[0] = insect_c;
                    clipped_cnt++;
                }
            }
        }
    }
}

//have to use __host__ __device__ here, could not recognize template???
#if NVCC_ON
__host__ __device__
#endif
inline void myswap(pt &a, pt &b)
{
	pt tmp = a;
	a = b;
	b = tmp;
}

#if NVCC_ON
__host__
#endif
inline void printTrgl(trgl t)
{
	cout<<"("<<t.p[0].x << ","<< t.p[1].x << "," << t.p[2].x << "," << t.p[0].x<<endl;
	cout<<"("<<t.p[0].y << ","<< t.p[1].y << "," << t.p[2].y << "," << t.p[0].y<<endl;
}

__host__ void GetResultToHost()
{
	hipError_t error;
	
	float *h_clipped_vert = (float*)malloc(mem_size_clipped_vert);
	error = hipMemcpy(h_clipped_vert, d_clipped_vert, mem_size_clipped_vert, hipMemcpyDeviceToHost);
	CudaSafeCall(error);

	int *h_clipped_n_vert = (int*)malloc(mem_size_clipped_n_vert);
	error = hipMemcpy(h_clipped_n_vert, d_clipped_n_vert, mem_size_clipped_n_vert, hipMemcpyDeviceToHost);
	CudaSafeCall(error);
}


#if NVCC_ON
__host__ __device__
#endif
void clip(trgl ts, trgl tc, pt clipped_array[6], int &clipped_cnt, instructSet *stateInstr)
{
	//mark inside or outside for the triangle vertices
	//and count the number of inside vertices
	int cnt_in_s = 0, cnt_in_c = 0;
	for(int i = 0; i < 3; i++)
	{
		if(tc.p[i].loc = testInside(tc.p[i], ts))
		   cnt_in_c++;

		if(ts.p[i].loc = testInside(ts.p[i], tc))
			cnt_in_s++;
	}

	//make the "in" vertices in the front of the array
	int a[3] = {0, 1, 0};
	for(int i = 0; i < 3; i++)
	{
		int idx = a[i];
		if(tc.p[idx].loc == 0 && tc.p[idx + 1].loc == 1)
			myswap(tc.p[idx], tc.p[idx + 1]);
		if(ts.p[idx].loc == 0 && ts.p[idx + 1].loc == 1)
			myswap(ts.p[idx], ts.p[idx + 1]);
	}

	bool test;
	if(1 == cnt_in_c && 1 == cnt_in_s)
		//test = BIntersectIncludeBoundary(ts.p[1], ts.p[2], tc.p[0], tc.p[1]);
		test = BIntersect(ts.p[1], ts.p[2], tc.p[0], tc.p[1]);

	int state = -1;
	if(0 == cnt_in_c && 0 == cnt_in_s)
		state = 0;
	else if(0 == cnt_in_c && 1 == cnt_in_s)
		state = 1;
	else if(1 == cnt_in_c && 0 == cnt_in_s)
		state = 2;
	else if(0 == cnt_in_c && 2 == cnt_in_s)
		state = 3;
	else if(2 == cnt_in_c && 0 == cnt_in_s)
		state = 4;
	else if(0 == cnt_in_c && 3 == cnt_in_s)
		state = 5;
	else if(3 == cnt_in_c && 0 == cnt_in_s)
		state = 6;
	else if(1 == cnt_in_c && 2 == cnt_in_s)
		state = 7;
	else if(2 == cnt_in_c && 1 == cnt_in_s)
		state = 8;
	else if(1 == cnt_in_c && 1 == cnt_in_s && !test)
		state = 9;
	else// if(1 == cnt_in_c && 1 == cnt_in_s && !test1) and (1 == cnt_in_c && 1 == cnt_in_s && test1 && test2)
		state = 10;
	//+cs

    instructSet is = stateInstr[state];
	if(is.doIns[0])//+sc
		AddIntersection(tc, ts, clipped_array, clipped_cnt);
	if(is.doIns[1])//+cs
		AddIntersection(ts, tc, clipped_array, clipped_cnt);
	if(is.doIns[12])
		clipped_array[clipped_cnt] = clipped_array[clipped_cnt - 1];
	if(is.doIns[2])//+c0-
		clipped_array[clipped_cnt - 1] = tc.p[0];
	if(is.doIns[3])//+s0-
		clipped_array[clipped_cnt - 1] = ts.p[0];
	if(is.doIns[13])
		clipped_cnt++;
	if(is.doIns[4])//+s0
		clipped_array[clipped_cnt++] = ts.p[0];
	if(is.doIns[5])//+c0
		clipped_array[clipped_cnt++] = tc.p[0];
	if(is.doIns[6])//+s1
		clipped_array[clipped_cnt++] = ts.p[1];
	if(is.doIns[7])//+c1
		clipped_array[clipped_cnt++] = tc.p[1];
	if(is.doIns[8])//+s2
		clipped_array[clipped_cnt++] = ts.p[2];
	if(is.doIns[9])//+c2
		clipped_array[clipped_cnt++] = tc.p[2];
	if(is.doIns[10])//+r0
		clipped_array[clipped_cnt++] = clipped_array[0];
	if(is.doIns[11])//+r0_s0
		clipped_array[0] = ts.p[0];


	//if number of edge less than 3, then this is not a polygon
	if(clipped_cnt > 0 && clipped_cnt < 3)
	{
	//	printTrgl(ts);
	//	printTrgl(tc);
	//	cout<<"state:"<<state<<endl;
	//	cout<<"clipped_cnt:"<<clipped_cnt<<endl;
		//cout<<"state:"<<state<<endl;
		//cout<<"clipped_cnt:"<<clipped_cnt<<endl;
		//cout<<"error:polygon has one or two vertices, impossible case!"<<endl;
		clipped_cnt = 0;
	//	exit(1);
	}

	//clipped_cnt = ts.p[0].x * 1000;//testInside(ts.p[0], tc);
//	clipped_array[0] = ts.p[0];
}


__global__ void clip_kernel(triangle *t_s, triangle *t_c, int2 *pair, int npair, polygon *clipped, int *clipped_n, instructSet *d_state)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= npair)
		return;

	triangle *t_s1 = &t_s[pair[idx].x];
	triangle *t_c1 = &t_c[pair[idx].y];

    trgl ts, tc;
    for(int i = 0; i < 3; i++)
    {
        ts.p[i].x = t_s1->p[i].x;
        ts.p[i].y = t_s1->p[i].y;
        tc.p[i].x = t_c1->p[i].x;
        tc.p[i].y = t_c1->p[i].y;
    }

	pt clipped_array[6];
	int clipped_cnt = 0;
	clip(ts, tc, clipped_array, clipped_cnt, d_state);
	//if(clipped_cnt > 6)
	//{
	//	clipped_cnt = 7;
	//}
	//
	for(int i = 0; i < clipped_cnt; i++)
	{
		clipped[idx].p[i].x = clipped_array[i].x;
		clipped[idx].p[i].y = clipped_array[i].y;
	}
	//if(clipped_cnt > 6)
	//	asm("trap;");
	clipped_n[idx] = clipped_cnt;
}


__host__
vector<point> clip_serial(triangle t_s, triangle t_c)
{
    vector<point> clipped;
    trgl ts, tc;
    for(int i = 0; i < 3; i++)
    {
        ts.p[i].x = t_s.p[i].x;
        ts.p[i].y = t_s.p[i].y;
        tc.p[i].x = t_c.p[i].x;
        tc.p[i].y = t_c.p[i].y;
    }
	pt clipped_array[6];
	int clipped_cnt = 0;
	clip(ts, tc, clipped_array, clipped_cnt, _stateSet);

    for(int i = 0; i < clipped_cnt; i++)
    {
        point p(clipped_array[i].x, clipped_array[i].y);
        clipped.push_back(p);
    }
    return clipped;
}

__host__ void finishCUDA()
{
	hipFree(d_clipped_n_vert);
	hipFree(d_clipped_vert);
	hipFree(d_trgl_s);
	hipFree(d_trgl_c);
	hipFree(d_pair);
	hipFree(d_state);
}

__host__ void initCUDA()
{
	int devID = 0;

	hipSetDevice(devID);

	hipError_t error;
	hipDeviceProp_t deviceProp;
	error = hipGetDevice(&devID);

	if (error != hipSuccess)
	{
		printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
	}

	error = hipGetDeviceProperties(&deviceProp, devID);

	if (deviceProp.computeMode == hipComputeModeProhibited)
	{
		fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
		exit(EXIT_SUCCESS);
	}

	if (error != hipSuccess)
	{
		printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
	}
	else
	{
		printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
	}
}

__host__ void loadDataToDevice(float* trgl_s, float* trgl_c, int ntrgl, int *pair, int npair)
{


    hipError_t error;
    unsigned int mem_size = ntrgl * 6 * sizeof(float);//3 vertices, each vertex has x and y(2 float)

    error = hipMalloc((void **) &d_trgl_s, mem_size);
    CudaSafeCall(error);

    error = hipMalloc((void **) &d_trgl_c, mem_size);
    CudaSafeCall(error);

    error = hipMemcpy(d_trgl_s, trgl_s, mem_size, hipMemcpyHostToDevice);
    CudaSafeCall(error);

    error = hipMemcpy(d_trgl_c, trgl_c, mem_size, hipMemcpyHostToDevice);
    CudaSafeCall(error);

    unsigned int mem_size_pair = npair * 2 * sizeof(int);

    error = hipMalloc((void **) &d_pair, mem_size_pair);
    CudaSafeCall(error);

    error = hipMemcpy(d_pair, pair, mem_size_pair, hipMemcpyHostToDevice);
    CudaSafeCall(error);

    //6 point * 2 value(x and y)
    mem_size_clipped_vert = npair * sizeof(polygon);

    error = hipMalloc((void **) &d_clipped_vert, mem_size_clipped_vert);
	CudaSafeCall(error);

	mem_size_clipped_n_vert = npair * sizeof(int);
	error = hipMalloc((void **) &d_clipped_n_vert, mem_size_clipped_n_vert);
	CudaSafeCall(error);

	//!!!!!!!!!!!!!!!!!!!!
	//assign space for stateSet and copy to device memory
	unsigned int mem_size_state = N_INSTR * N_STATE * sizeof(bool);
	error = hipMalloc((void **) &d_state, mem_size_state);
	error = hipMemcpy(d_state, _stateSet, mem_size_state, hipMemcpyHostToDevice);
    CudaSafeCall(error);


	_npair = npair;

}


__global__ void gen_cells_kernel(vtkIdType* cellArray, int N, int* preSum, int* nVert)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= N)
		return;
	
	int begin = idx + preSum[idx];
	int num = nVert[idx];
	int input = preSum[idx];
	cellArray[begin++] = num;
	for(int i = 0; i < num; i++)
		cellArray[begin++] = input++;
}

__global__ void gen_points_kernel(float3 *points, polygon *clipped_vert, int *preSum, int *nVert, int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= N)
		return;

	int num = nVert[idx];
	int begin = preSum[idx];
	for(int i = 0; i < num; i++, begin++)
	{
		points[begin].x = clipped_vert[idx].p[i].x;
		points[begin].y = clipped_vert[idx].p[i].y;
		points[begin].z = 0;
	}
}

template <typename T>
__host__ void printArray(T *d_array, int size, int num, bool front)
{
	unsigned int mem_size = size * sizeof(T);
	T *h_array;
	h_array = (T*)malloc(mem_size);
	hipError_t error = hipMemcpy(h_array, d_array, mem_size, hipMemcpyDeviceToHost);
    CudaSafeCall(error);
	cout<< "print array:"<<endl;
	for(int i = 0; i < num; i++)
	{
		if(front)
			cout<<h_array[i]<<endl;
		else
			cout<<h_array[size - 1 - i]<<endl;
	}
}

__host__ void printTriangle(triangle* d_trgl, int i)
{
	unsigned int mem_size = sizeof(triangle);
	triangle* h_trgl;
	h_trgl = (triangle*)malloc(mem_size);
	hipError_t error = hipMemcpy(h_trgl, d_trgl + i, mem_size, hipMemcpyDeviceToHost);
    CudaCheckError();
	cout<< "print Triangle:"<<endl;
	for(int j = 0; j < 3; j++)
		cout<<h_trgl->p[j].x<<","<<h_trgl->p[j].y<<endl;
	free(h_trgl);
}



__host__ void printPair(int2 *d_array, int size, int idx)
{
	unsigned int mem_size = size * sizeof(int2);
	int2 *h_array;
	h_array = (int2*)malloc(mem_size);
	hipError_t error = hipMemcpy(h_array, d_array, mem_size, hipMemcpyDeviceToHost);
    CudaSafeCall(error);
	cout<< "print printPair:"<<endl;
	cout<<h_array[idx].x<<","<<h_array[idx].y<<endl;
}

__host__ void printPolygon(polygon *d_array, int size, int idx)
{
	unsigned int mem_size = size * sizeof(polygon);
	polygon *h_array;
	h_array = (polygon*)malloc(mem_size);
	hipError_t error = hipMemcpy(h_array, d_array, mem_size, hipMemcpyDeviceToHost);
    CudaSafeCall(error);
	cout<< "print polygon:"<<endl;
	for(int i = 0; i < 6; i++)
	{
		cout<< h_array[idx].p[i].x << "," << h_array[idx].p[i].y  <<endl;
	}
}



template <typename T>
__host__ void checkArray(T *d_array, int size)
{
	unsigned int mem_size = size * sizeof(T);
	T *h_array;
	h_array = (T*)malloc(mem_size);
	hipError_t error = hipMemcpy(h_array, d_array, mem_size, hipMemcpyDeviceToHost);
    CudaSafeCall(error);
	for(int i = 0; i < size; i++)
	{
		if(h_array[i] > 6)
			cout<<"check:"<<i<<","<<h_array[i]<<endl;
	}
}

__host__
void runKernel(float* &points, vtkIdType* &cells, int &nCells, int &nPts, int nBlock)//triangle *t_s, triangle *t_c, int2 *pair, int npair)//, polygon *clipped, int *clipped_n)
{
	dim3 block(nBlock, 1, 1);
    dim3 grid(ceil((float)_npair / block.x), 1, 1);

	
	//printTriangle((triangle*)d_trgl_s, 16546);
	//printTriangle((triangle*)d_trgl_c, 88008);
	
	clip_kernel<<<grid, block>>>
		((triangle*)d_trgl_s, (triangle*)d_trgl_c, 
		(int2*)d_pair, _npair, 
		d_clipped_vert, d_clipped_n_vert,
		d_state);
	CudaCheckError();



	//printPair(d_pair, _npair, 681046);
	//printPolygon(d_clipped_vert, _npair, 681046);


	//checkArray<int>(d_clipped_n_vert, _npair);

	hipError_t error;

	int* d_preSum;
    error = hipMalloc((void **) &d_preSum, mem_size_clipped_n_vert);
    CudaSafeCall(error);

	//previous sum for the number of vertices
	thrust::device_ptr<int> d_ptr_clipped_n_vert(d_clipped_n_vert);
	//cout<<"num of vert:"<<d_ptr_clipped_n_vert[681046]<<endl;
	thrust::device_ptr<int> d_ptr_clipped_preSum(d_preSum);
	thrust::exclusive_scan(thrust::device, d_ptr_clipped_n_vert, d_ptr_clipped_n_vert + _npair, d_ptr_clipped_preSum); 

	nPts = d_ptr_clipped_n_vert[_npair - 1] + d_ptr_clipped_preSum[_npair - 1];
	//cout<<"nPts:"<<nPts<<endl;
	
	///////////points
	float3* d_points;
	unsigned int mem_size_points = nPts * sizeof(float3);
	error = hipMalloc((void **) &d_points, mem_size_points);
	gen_points_kernel<<<grid, block>>>(d_points, d_clipped_vert, d_preSum, d_clipped_n_vert, _npair);

	float3* h_points = (float3*)malloc(mem_size_points);
	error = hipMemcpy(h_points, d_points, mem_size_points, hipMemcpyDeviceToHost);

	//////cells//////
	thrust::device_ptr<int> d_ptr_clipped_n_vert_end = thrust::remove(thrust::device, d_ptr_clipped_n_vert, d_ptr_clipped_n_vert + _npair, 0);
	nCells = d_ptr_clipped_n_vert_end - d_ptr_clipped_n_vert;

	int* d_preSum_compact;
	unsigned int mem_size_preSum_compact = nCells * sizeof(int);
    hipMalloc((void **) &d_preSum_compact, mem_size_preSum_compact);
	thrust::device_ptr<int> d_ptr_clipped_preSum_compact(d_preSum_compact);
	thrust::exclusive_scan(thrust::device, d_ptr_clipped_n_vert, d_ptr_clipped_n_vert + nCells, d_ptr_clipped_preSum_compact);
	//cout<<"nCells:"<<nCells<<endl;

	
	//cout<<"d_ptr_preSum_compact:"<<endl;
	//for(int i = 0; i < 10; i++)
	//	cout<<d_ptr_clipped_preSum_compact[i]<<endl;

	int size_cells = nPts + nCells;

	unsigned int mem_size_cells = size_cells * sizeof(vtkIdType);


	
	//size_t fr, ttl;
	//hipMemGetInfo(&fr, &ttl);
	//cout<<"fr:"<<fr<<endl;
	//cout<<"ttl:"<<ttl<<endl;

	vtkIdType* d_cells;
    error = hipMalloc((void **) &d_cells, mem_size_cells);
	CudaSafeCall( error );

	dim3 block2(nBlock, 1, 1);
    dim3 grid2(ceil((float)size_cells / block2.x), 1, 1);
	

	cout<<"grid2:"<<grid2.x<<","<<grid2.y<<","<<grid2.z<<endl;
	gen_cells_kernel<<<grid2, block2>>>(d_cells, nCells, d_preSum_compact, d_clipped_n_vert);
	//printArray<vtkIdType>(d_cells, 100, 10, true);
	//printArray<int>(d_preSum_compact, nCells, 10, false);
	//printArray<int>(d_clipped_n_vert, nCells, 10, false);

	vtkIdType* h_cells = (vtkIdType*)malloc(mem_size_cells);
	error = hipMemcpy(h_cells, d_cells, mem_size_cells, hipMemcpyDeviceToHost);
	hipFree(d_cells);
	hipFree(d_clipped_n_vert);
	hipFree(d_preSum);
	hipFree(d_points);
	
	hipFree(d_trgl_s);
	hipFree(d_trgl_c);
	hipFree(d_pair);
	hipFree(d_state);
	hipFree(d_preSum_compact);

	points = (float*)h_points;
	cells = h_cells;
}