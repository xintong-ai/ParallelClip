#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
#include "stdlib.h"
#include "stdio.h"
#define EPS 0.00001

struct instructSet
{
    bool doIns[14];
};


__constant__ instructSet STATE_SET[11];

template<typename T>
__device__ inline void swap(T a, T b)
{
	T tmp = a;
	a = b;
    b = tmp;
}


struct pt
{
    float x;
    float y;
    float loc;
    __device__
    pt(float _x, float _y)
    {
        x = _x;
        y = _y;
        loc = -1;
    }
    __device__
    pt()
    {
        loc = -1;
    }
};

struct trgl
{
    //the first 3 points are the vertex
    //others are reserved forintersection points
    pt p[9];
};

struct point
{
    float x;
    float y;
    point(float _x, float _y)
    {
        x = _x;
        y = _y;
    }
    point(){};
};

struct triangle
{
    point p[3];
    __device__
    triangle(point p0, point p1, point p2)
    {
        p[0] = p0;
        p[1] = p1;
        p[2] = p2;
    }
    triangle(point _p[4])
    {
        p[0] = _p[0];
        p[1] = _p[1];
        p[2] = _p[2];
    }
};





__host__
void setStateInstr()
{

    instructSet stateSet[11];
    for(int s = 0; s < 11; s++)
        for(int i = 0; i < 14; i++)
            stateSet[s].doIns[i] = false;

    stateSet[0].doIns[1] = true;

    stateSet[1].doIns[0] = true;
    stateSet[1].doIns[4] = true;

    stateSet[2].doIns[1] = true;
    stateSet[2].doIns[5] = true;

    stateSet[3].doIns[0] = true;
    stateSet[3].doIns[4] = true;
    stateSet[3].doIns[6] = true;

    stateSet[4].doIns[1] = true;
    stateSet[4].doIns[5] = true;
    stateSet[4].doIns[7] = true;

    stateSet[5].doIns[4] = true;
    stateSet[5].doIns[6] = true;
    stateSet[5].doIns[8] = true;

    stateSet[6].doIns[5] = true;
    stateSet[6].doIns[7] = true;
    stateSet[6].doIns[9] = true;

    stateSet[7].doIns[0] = true;
    stateSet[7].doIns[12] = true;
    stateSet[7].doIns[2] = true;
    stateSet[7].doIns[13] = true;
    stateSet[7].doIns[4] = true;
    stateSet[7].doIns[6] = true;

    stateSet[8].doIns[1] = true;
    stateSet[8].doIns[12] = true;
    stateSet[8].doIns[3] = true;
    stateSet[8].doIns[13] = true;
    stateSet[8].doIns[5] = true;
    stateSet[8].doIns[7] = true;

    stateSet[9].doIns[1] = true;
    stateSet[9].doIns[5] = true;
    stateSet[9].doIns[10] = true;
    stateSet[9].doIns[11] = true;

    stateSet[10].doIns[1] = true;
    stateSet[10].doIns[3] = true;
    stateSet[10].doIns[5] = true;

    hipMemcpyToSymbol(HIP_SYMBOL(STATE_SET),
                       &stateSet,
                       14 * 11 *sizeof(bool),
                       0,
                       hipMemcpyHostToDevice);
}


template<typename T>
__device__ inline T min(T x1, T x2, T x3)
{
    T xmin;
    if(x1 < x2)
        xmin = x1;
    else
        xmin = x2;

    if(x2 < xmin)
        xmin = x2;
    if(x3 < xmin)
        xmin = x3;

    return xmin;
}


template<typename T>
__device__ inline T max(T x1, T x2, T x3)
{
    T xmax;
    if(x1 > x2)
        xmax = x1;
    else
        xmax = x2;

    if(x2 > xmax)
        xmax = x2;
    if(x3 > xmax)
        xmax = x3;

    return xmax;
}

inline void CheckError(hipError_t error)
{
	if (error != hipSuccess)
	{
		printf("returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}
}

extern "C"
__host__ void initCUDA()
{
	int devID = 0;

//	if (checkCmdLineFlag(argc, (const char **)argv, "device"))
//	{
		//devID = getCmdLineArgumentInt(argc, (const char **)argv, "device");
		hipSetDevice(devID);
//	}

	hipError_t error;
	hipDeviceProp_t deviceProp;
	error = hipGetDevice(&devID);

	if (error != hipSuccess)
	{
		printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
	}

	error = hipGetDeviceProperties(&deviceProp, devID);

	if (deviceProp.computeMode == hipComputeModeProhibited)
	{
		fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
		exit(EXIT_SUCCESS);
	}

	if (error != hipSuccess)
	{
		printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
	}
	else
	{
		printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
	}
}

extern "C"
__host__ void loadDataToDevice(float* trgl_s, float* trgl_c, int ntrgl, int *pair, int npar)
{
    float *d_trgl_s;
    float *d_trgl_c;
    int2 *d_pair;
    float *d_clipped_vert;
    float *d_clipped_n_vert;

    hipError_t error;
    unsigned int mem_size = ntrgl * 6 * sizeof(float);//3 vertices, each vertex has x and y(2 float)

    error = hipMalloc((void **) &d_trgl_s, mem_size);
    CheckError(error);

    error = hipMalloc((void **) &d_trgl_c, mem_size);
    CheckError(error);

    error = hipMemcpy(d_trgl_s, trgl_s, mem_size, hipMemcpyHostToDevice);
    CheckError(error);

    error = hipMemcpy(d_trgl_c, trgl_c, mem_size, hipMemcpyHostToDevice);
    CheckError(error);

    unsigned int mem_size_pair = npar * 2 * sizeof(int);

    error = hipMalloc((void **) &d_pair, mem_size_pair);
    CheckError(error);

    error = hipMemcpy(d_pair, pair, mem_size_pair, hipMemcpyHostToDevice);
    CheckError(error);

    //6 point * 2 value(x and y)
    unsigned int mem_size_clipped_vert = npar * 12 * sizeof(float);

    error = hipMalloc((void **) &d_clipped_vert, mem_size_clipped_vert);
	CheckError(error);

	unsigned int mem_size_clipped_n_vert = npar * sizeof(int);
	error = hipMalloc((void **) &d_clipped_n_vert, mem_size_clipped_n_vert);
	CheckError(error);

    setStateInstr();

}


__device__
inline bool BIntersect(pt p1, pt p2, pt q1, pt q2)
{
  float  tp, tq, par;

  par = (float) ((p2.x - p1.x)*(q2.y - q1.y) -
                 (p2.y - p1.y)*(q2.x - q1.x));

  if (!par) return 0;                               /* parallel lines */
  tp = ((q1.x - p1.x)*(q2.y - q1.y) - (q1.y - p1.y)*(q2.x - q1.x))/par;
  tq = ((p2.y - p1.y)*(q1.x - p1.x) - (p2.x - p1.x)*(q1.y - p1.y))/par;

  //touching the boundary is not inside
  if(tp<=0 || tp>=1 || tq<=0 || tq>=1) return 0;

  return 1;
}

//touching boundary is also intersect
__device__
inline bool BIntersectIncludeBoundary(pt p1, pt p2, pt q1, pt q2)
{
  float  tp, tq, par;

  par = (float) ((p2.x - p1.x)*(q2.y - q1.y) -
                 (p2.y - p1.y)*(q2.x - q1.x));

  if (!par) return 0;                               /* parallel lines */

  tp = ((q1.x - p1.x)*(q2.y - q1.y) - (q1.y - p1.y)*(q2.x - q1.x))/par;
  if(tp<0 || tp>1 )
      return 0;

  tq = ((p2.y - p1.y)*(q1.x - p1.x) - (p2.x - p1.x)*(q1.y - p1.y))/par;
  //touching the boundary is not inside
  if(tq<0 || tq>1)
      return 0;

  return 1;
}

__device__
inline void Intersect(pt p1, pt p2, pt q1, pt q2,
        pt &pi, pt &qi)
{
    float tp, tq, par;

    par = (float) ((p2.x - p1.x)*(q2.y - q1.y) -
                   (p2.y - p1.y)*(q2.x - q1.x));

    if (!par)
        return;                               /* parallel lines */

    tp = ((q1.x - p1.x)*(q2.y - q1.y) - (q1.y - p1.y)*(q2.x - q1.x))/par;
    tq = ((p2.y - p1.y)*(q1.x - p1.x) - (p2.x - p1.x)*(q1.y - p1.y))/par;

    if(tp<0 || tp>1 || tq<0 || tq>1)
        return;

//    pi.in = true;
//    qi.in = true;
    pi.x = p1.x + tp*(p2.x - p1.x);
    pi.y = p1.y + tp*(p2.y - p1.y);
    qi.x = pi.x;
    qi.y = pi.y;

    //this can be replaced with tp and tq with care
    pi.loc = tp;// dist(p1.x, p1.y, x, y) / dist(p1.x, p1.y, p2.x, p2.y);
    qi.loc = tq;// dist(q1.x, q1.y, x, y) / dist(q1.x, q1.y, q2.x, q2.y);
}

__device__
inline bool testInside(pt p, trgl t)
{
    bool inside = false;
    pt left( -999, p.y);//create(0, point->y, 0, 0, 0, 0, 0, 0, 0, 0.);
    for(int i = 0; i < 3; i++)
    {
        if(BIntersect(left, p, t.p[i], t.p[(i+1)%3]))
            inside = !inside;
    }
    return inside;
}

__device__
inline void AddIntersection(trgl ts, trgl tc, pt *clipped_array, int &clipped_cnt)
{
    for(int ic = 0; ic < 3; ic++)
    {
        for(int is = 0; is < 3; is++)
        {
            pt insect_s, insect_c;
            Intersect(tc.p[ic], tc.p[(ic+1)%3], ts.p[is], ts.p[(is+1)%3 ],
                    insect_c, insect_s);

            if(insect_c.loc >= 0)
            {
                insect_c.loc += ic;
                if(clipped_cnt > 0)
                {
                    if(insect_c.loc > clipped_array[clipped_cnt - 1].loc)
                        clipped_array[clipped_cnt++] = insect_c;
                    else if(insect_c.loc < clipped_array[clipped_cnt - 1].loc)
                    {
                        clipped_array[clipped_cnt] = clipped_array[clipped_cnt - 1];
                        clipped_array[clipped_cnt - 1] = insect_c;
                        clipped_cnt++;
                    }
                    //else :insect_c.loc == clipped_vert[isect_cnt - 1].loc
                    //don't add anything
                }
                else
                {
                    clipped_array[0] = insect_c;
                    clipped_cnt++;
                }
            }
        }
    }
}

//line(p1, p2) is parallel with line(q1, q2)
__device__
inline bool parallel(pt p1, pt p2, pt q1, pt q2)
{
  float par = (float) ((p2.x - p1.x)*(q2.y - q1.y) -
                 (p2.y - p1.y)*(q2.x - q1.x));
  if(abs(par)<EPS)
      return true;
  else
      return false;
}

struct polygon
{
	point p[6];
};

__global__ void clip_kernel(triangle *t_s, triangle *t_c, int2 *pair, int npair, polygon *clipped, int *clipped_n)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= npair)
		return;

	triangle *t_s1 = &t_s[pair[idx].x];
	triangle *t_c1 = &t_c[pair[idx].y];

    trgl ts, tc;
    for(int i = 0; i < 3; i++)
    {
        ts.p[i].x = t_s1->p[i].x;
        ts.p[i].y = t_s1->p[i].y;
        tc.p[i].x = t_c1->p[i].x;
        tc.p[i].y = t_c1->p[i].y;
    }

	float sx[2], sy[2], cx[2], cy[2];
    sx[0] = min<float>(ts.p[0].x, ts.p[1].x, ts.p[2].x);
	cx[1] = max<float>(tc.p[0].x, tc.p[1].x, tc.p[2].x);
	if(sx[0] >= cx[1])
	{
		clipped_n[idx] = 0;
		return;
	}

	sy[0] = min<float>(ts.p[0].y, ts.p[1].y, ts.p[2].y);
	cy[1] = max<float>(tc.p[0].y, tc.p[1].y, tc.p[2].y);
	if(sy[0] >= cy[1])
	{
		clipped_n[idx] = 0;
		return;
	}

	cx[0] = min<float>(tc.p[0].x, tc.p[1].x, tc.p[2].x);
	sx[1] = max<float>(ts.p[0].x, ts.p[1].x, ts.p[2].x);
	if(cx[0] >= sx[1])
	{
		clipped_n[idx] = 0;
		return;
	}

	cy[0] = min<float>(tc.p[0].y, tc.p[1].y, tc.p[2].y);
	sy[1] = max<float>(ts.p[0].y, ts.p[1].y, ts.p[2].y);
	if(cy[0] >= sy[1])
	{
		clipped_n[idx] = 0;
		return;
	}


	//mark inside or outside for the triangle vertices
	//and count the number of inside vertices
	int cnt_in_s = 0, cnt_in_c = 0;
	for(int i = 0; i < 3; i++)
	{
		if(tc.p[i].loc = testInside(tc.p[i], ts))
		   cnt_in_c++;

		if(ts.p[i].loc = testInside(ts.p[i], tc))
			cnt_in_s++;
	}

	//make the "in" vertices in the front of the array
	int a[3] = {0, 1, 0};
	for(int i = 0; i < 3; i++)
	{
		int idx = a[i];
		if(!tc.p[idx].loc && tc.p[idx + 1].loc)
			swap(tc.p[idx], tc.p[idx + 1]);
		if(!ts.p[idx].loc && ts.p[idx + 1].loc)
			swap(ts.p[idx], ts.p[idx + 1]);
	}

	bool test;
	if(1 == cnt_in_c && 1 == cnt_in_s)
		test = BIntersectIncludeBoundary(ts.p[1], ts.p[2], tc.p[0], tc.p[1]);

	int state = -1;
	if(0 == cnt_in_c && 0 == cnt_in_s)
		state = 0;
	else if(0 == cnt_in_c && 1 == cnt_in_s)
		state = 1;
	else if(1 == cnt_in_c && 0 == cnt_in_s)
		state = 2;
	else if(0 == cnt_in_c && 2 == cnt_in_s)
		state = 3;
	else if(2 == cnt_in_c && 0 == cnt_in_s)
		state = 4;
	else if(0 == cnt_in_c && 3 == cnt_in_s)
		state = 5;
	else if(3 == cnt_in_c && 0 == cnt_in_s)
		state = 6;
	else if(1 == cnt_in_c && 2 == cnt_in_s)
		state = 7;
	else if(2 == cnt_in_c && 1 == cnt_in_s)
		state = 8;
	else if(1 == cnt_in_c && 1 == cnt_in_s && !test)
		state = 9;
	else// if(1 == cnt_in_c && 1 == cnt_in_s && !test1) and (1 == cnt_in_c && 1 == cnt_in_s && test1 && test2)
		state = 10;
	//+cs

	pt clipped_array[6];

	int clipped_cnt = 0;
    instructSet is = STATE_SET[state];
	if(is.doIns[0])//+sc
		AddIntersection(tc, ts, clipped_array, clipped_cnt);
	if(is.doIns[1])//+cs
		AddIntersection(ts, tc, clipped_array, clipped_cnt);
	if(is.doIns[12])
		clipped_array[clipped_cnt] = clipped_array[clipped_cnt - 1];
	if(is.doIns[2])//+c0-
		clipped_array[clipped_cnt - 1] = tc.p[0];
	if(is.doIns[3])//+s0-
		clipped_array[clipped_cnt - 1] = ts.p[0];
	if(is.doIns[13])
		clipped_cnt++;
	if(is.doIns[4])//+s0
		clipped_array[clipped_cnt++] = ts.p[0];
	if(is.doIns[5])//+c0
		clipped_array[clipped_cnt++] = tc.p[0];
	if(is.doIns[6])//+s1
		clipped_array[clipped_cnt++] = ts.p[1];
	if(is.doIns[7])//+c1
		clipped_array[clipped_cnt++] = tc.p[1];
	if(is.doIns[8])//+s2
		clipped_array[clipped_cnt++] = ts.p[2];
	if(is.doIns[9])//+c2
		clipped_array[clipped_cnt++] = tc.p[2];
	if(is.doIns[10])//+r0
		clipped_array[clipped_cnt++] = clipped_array[0];
	if(is.doIns[11])//+r0_s0
		clipped_array[0] = ts.p[0];

	for(int i = 0; i < clipped_cnt; i++)
	{
		clipped[idx].p[i].x = clipped_array[i].x;
		clipped[idx].p[i].y = clipped_array[i].y;
	}
	clipped_n[idx] = clipped_cnt;
}


